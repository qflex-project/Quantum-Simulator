#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "common.h"
#include "gpu.h"
#include "pt.h"

#define M_RANGE 512
#define M_PREC 10000
#define OPS_BLOCK 300  // change on compilation

bool error();
static int inst = 0;
static int call_count = 0;
static int call_peer_count = 0;

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

struct DEV_OP {
  long arg[TAM_ARG];
  hipFloatComplex matrix[4];
};

extern "C" bool setDevice(int num = 0) { return hipFree(0); }

extern "C" bool enablePeerAccess() {
  hipSetDevice(0);
  hipDeviceEnablePeerAccess(1, 0);

  hipSetDevice(1);
  hipDeviceEnablePeerAccess(0, 0);

  hipGetLastError();

  return true;
}

__constant__ long c_arg[1][1];
__constant__ hipFloatComplex cmatrix[1][1];

__constant__ DEV_OP op[OPS_BLOCK];

static hipFloatComplex* gpu_mem[4];
__constant__ hipFloatComplex* gpu_pointer[4];

inline int GET_BLOCK_ID(PT* pt, int coalesc, int qbs_region) {
  return (pt->end - coalesc) / (qbs_region - coalesc);
}

__device__ long OPEN_SPACE(long value, int from_bit, int n) {
  return ((value >> from_bit) << (from_bit + n)) |
         (value & ((1 << from_bit) - 1));
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// extern "C"
template <int t_TAM_BLOCK, int t_REPT, int t_COALESC>
__global__ void ApplyValuesC01(int const b_pos, int const n_bits,
                               int const count, int const rept_bits,
                               int const shift, int const block_shift) {
  long p, g_pos1, g_pos2, block = (blockIdx.x + block_shift);

  int i, c, thId = threadIdx.x;

  __shared__ hipFloatComplex s[t_REPT * t_TAM_BLOCK * 2];

  long block_base;

  block_base = block << t_COALESC;
  block_base = OPEN_SPACE(block_base, b_pos, n_bits);

  long g_pos2_mask = (1 << (b_pos + n_bits - rept_bits - 1));

  // copy amplitudes from global memory to shared memory
  for (i = 0; i < t_REPT; i++) {
    p = thId + i * t_TAM_BLOCK * 2;  // another start

    g_pos1 =
        block_base | ((p >> t_COALESC) << b_pos) | (p & ((1 << t_COALESC) - 1));
    g_pos2 = g_pos1 | g_pos2_mask;

    s[p] = gpu_pointer[g_pos1 / shift][g_pos1 % shift];
    s[p + t_TAM_BLOCK] = gpu_pointer[g_pos2 / shift][g_pos2 % shift];
  }

  int pos0, pos1, op_bit;
  hipFloatComplex tmp;

  // compute the operators for the amplitudes on the shared memory
  for (c = 0; c < count; c++) {
    __syncthreads();

    op_bit = 1 << op[c].arg[SHIFT];

    if (((block_base & op[c].arg[CTRL_MASK]) == op[c].arg[CTRL_VALUE])) {
      for (i = 0; i < t_REPT; i++) {
        p = thId + i * t_TAM_BLOCK;

        pos0 = (p * 2) - (p & (op_bit - 1));
        pos1 = pos0 | op_bit;
        if ((pos0 & op[c].arg[CTRL_REG_MASK]) == op[c].arg[CTRL_REG_VALUE]) {
          tmp = hipCaddf(hipCmulf(s[pos0], op[c].matrix[0]),
                        hipCmulf(s[pos1], op[c].matrix[1]));
          s[pos1] = hipCaddf(hipCmulf(s[pos0], op[c].matrix[2]),
                            hipCmulf(s[pos1], op[c].matrix[3]));
          s[pos0] = tmp;
        }
      }
    }
  }
  __syncthreads();

  // copy results from shared memory to global memory
  for (i = 0; i < t_REPT; i++) {
    p = thId + i * t_TAM_BLOCK * 2;  // another start

    g_pos1 =
        block_base | ((p >> t_COALESC) << b_pos) | (p & ((1 << t_COALESC) - 1));
    g_pos2 = g_pos1 | g_pos2_mask;

    gpu_pointer[g_pos1 / shift][g_pos1 % shift] = s[p];
    gpu_pointer[g_pos2 / shift][g_pos2 % shift] = s[p + t_TAM_BLOCK];
  }
}

// Kernel para execução com múltiplas GPUs se comunicando usando DMA (Direct
// Memory Access)
template <int t_TAM_BLOCK, int t_REPT, int t_COALESC>
void GpuExecution01(float complex* state, PT** pts, int qubits, int qbs_region,
                    int multi_gpu, int num_it) {
  DEV_OP operators[OPS_BLOCK];

  inst = 0;

  dim3 block, dim;

  long mem_size = pow(2.0, qubits);
  long mem_desloc = mem_size / multi_gpu;

  int rept_bits = (int)log2((float)t_REPT);

  long nth = mem_size / multi_gpu / t_REPT /
             2;  // /2 porque cada thread fica responsável por duas posições &
                 // /2 pelas 2 GPUS

  long malloc_size = (mem_size * (sizeof(float complex))) / multi_gpu;

  block.x = t_TAM_BLOCK;
  (nth > block.x) ? dim.x = nth / block.x : block.x = nth;

  int block_region_size = qbs_region;

  if (block_region_size < qbs_region) {
    printf("ERRO: Região do bloco menor que a região de qubits\n");
    exit(1);
  }

  if (multi_gpu > 1) {
    for (int d = 0; d < multi_gpu; d++) {
      hipSetDevice(d);
      for (int j = 0; j < multi_gpu; j++)
        if (d != j) hipDeviceEnablePeerAccess(j, 0);
    }
    hipGetLastError();
  }

  // NULL state means it should already be on the gpu's memory (projection)
  if (state != NULL) {
    for (int d = 0; d < multi_gpu; d++) {
      hipSetDevice(d);
      hipMalloc(&gpu_mem[d], malloc_size);
      error();
      hipMemcpy(gpu_mem[d], state + mem_desloc * d, malloc_size,
                 hipMemcpyHostToDevice);
      error();
    }
    for (int d = 0; d < multi_gpu; d++) {
      hipSetDevice(d);
      hipMemcpyToSymbol(HIP_SYMBOL(gpu_pointer), gpu_mem,
                         multi_gpu * sizeof(hipFloatComplex*));
      error();
    }
  }

  int i;
  for (int it = 0; it < num_it; it++) {
    i = 0;

    while (pts[i] != NULL) {
      int region_start, is_peer,
          c = 0;  //, qbs_block_id, max_end // atualmente não utilizados
      is_peer = 0;

      while (pts[i + c] != NULL && pts[i + c]->end < t_COALESC &&
             c < OPS_BLOCK) {
        c++;
      }

      // max_end = t_COALESC; // atualmente não utilizada

      int s_max, s_min = t_COALESC;

      int extra_region = (block_region_size - t_COALESC);

      if (pts[i + c] != NULL && c < OPS_BLOCK) {
        s_min = s_max = pts[i + c]->end;

        do {
          int e = pts[i + c]->end;
          if (e < t_COALESC) {
          } else if ((e >= s_min) && ((e - s_min) < extra_region)) {
            s_max = max(s_max, e);
          } else if ((e <= s_max) && ((s_max - e) < extra_region)) {
            s_min = min(s_min, e);
          } else {
            break;
          }

          c++;
        } while (pts[i + c] != NULL && c < OPS_BLOCK);
      }
      region_start = max(t_COALESC, s_max - extra_region + 1);

      is_peer = ((region_start + (block_region_size - t_COALESC)) >
                 (qubits - multi_gpu + 1));

      for (int j = 0; j < c; j++) {
        memcpy(operators[j].matrix, pts[i + j]->matrix,
               4 * sizeof(float complex));
        error();
        pts[i + j]->setArgsGPU(operators[j].arg, region_start,
                               block_region_size, t_COALESC);
      }

      if (is_peer) {
        for (int d = 0; d < multi_gpu; d++) {
          hipSetDevice(d);
          hipDeviceSynchronize();
        }
      }

      for (int d = 0; d < multi_gpu; d++) {
        hipSetDevice(d);
        error();
        hipMemcpyToSymbol(HIP_SYMBOL(op), operators, c * sizeof(DEV_OP));
        error();
      }

      for (int d = 0; d < multi_gpu; d++) {
        hipSetDevice(d);
        error();
        ApplyValuesC01<t_TAM_BLOCK, t_REPT, t_COALESC><<<dim, block>>>(
            region_start, extra_region, c, rept_bits, mem_desloc, dim.x * d);
        error();
      }
      hipDeviceSynchronize();
      error();

      for (int d = 0; d < multi_gpu; d++) {
        hipSetDevice(d);
        error();
        hipDeviceSynchronize();
        error();
      }

      call_count++;

      if (is_peer) call_peer_count++;

      i += c;
    }
  }

  if (state != NULL) {
    for (int d = 0; d < multi_gpu; d++) {
      hipMemcpy(state + mem_desloc * d, gpu_mem[d], malloc_size,
                 hipMemcpyDeviceToHost);
      error();
      hipFree(gpu_mem[d]);
      error();
    }
  }
}

// Segundo Wrapper -- tamanho de bloco e número de projeções por bloco
template <int t_COALESC>
void GEWrapper2(float complex* state, PT** pts, int qubits, int qbs_region,
                int multi_gpu, int tam_block, int rept, int num_it) {
  switch (tam_block) {
    case 32:
      switch (rept) {
        case 1:
          GpuExecution01<32, 1, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 2:
          GpuExecution01<32, 2, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 4:
          GpuExecution01<32, 4, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 8:
          GpuExecution01<32, 8, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 16:
          GpuExecution01<32, 16, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 32:
          GpuExecution01<32, 32, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        default:
          printf("Invalid REPT");
      }
      break;
    case 64:
      switch (rept) {
        case 1:
          GpuExecution01<64, 1, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 2:
          GpuExecution01<64, 2, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 4:
          GpuExecution01<64, 4, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 8:
          GpuExecution01<64, 8, t_COALESC>(state, pts, qubits, qbs_region,
                                           multi_gpu, num_it);
          break;
        case 16:
          GpuExecution01<64, 16, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 32:
          GpuExecution01<64, 32, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        default:
          printf("Invalid REPT");
      }
      break;
    case 128:
      switch (rept) {
        case 1:
          GpuExecution01<128, 1, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 2:
          GpuExecution01<128, 2, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 4:
          GpuExecution01<128, 4, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 8:
          GpuExecution01<128, 8, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 16:
          GpuExecution01<128, 16, t_COALESC>(state, pts, qubits, qbs_region,
                                             multi_gpu, num_it);
          break;
        default:
          printf("Invalid REPT");
      }
      break;
    case 256:
      switch (rept) {
        case 1:
          GpuExecution01<256, 1, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 2:
          GpuExecution01<256, 2, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 4:
          GpuExecution01<256, 4, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 8:
          GpuExecution01<256, 8, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        default:
          printf("Invalid REPT");
      }
      break;
    case 512:
      switch (rept) {
        case 1:
          GpuExecution01<512, 1, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 2:
          GpuExecution01<512, 2, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        case 4:
          GpuExecution01<512, 4, t_COALESC>(state, pts, qubits, qbs_region,
                                            multi_gpu, num_it);
          break;
        default:
          printf("Invalid REPT");
      }
      break;
    case 1024:
      switch (rept) {
        case 1:
          GpuExecution01<1024, 1, t_COALESC>(state, pts, qubits, qbs_region,
                                             multi_gpu, num_it);
          break;
        case 2:
          GpuExecution01<1024, 2, t_COALESC>(state, pts, qubits, qbs_region,
                                             multi_gpu, num_it);
          break;
        default:
          printf("Invalid REPT");
      }
      break;
    default:
      printf("Invalid TAM_BLOCK");
  }
}

// Primeiro Wrapper -- Coalescimento
extern "C" float complex* GpuExecutionWrapper(float complex* state, PT** pts,
                                              int qubits, int coalesc,
                                              int qbs_region, int multi_gpu,
                                              int tam_block, int rept,
                                              int num_it) {
  switch (coalesc) {
    case 0:
      GEWrapper2<0>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 1:
      GEWrapper2<1>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 2:
      GEWrapper2<2>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 3:
      GEWrapper2<3>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 4:
      GEWrapper2<4>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 5:
      GEWrapper2<5>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 6:
      GEWrapper2<6>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 7:
      GEWrapper2<7>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 8:
      GEWrapper2<8>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    case 9:
      GEWrapper2<9>(state, pts, qubits, qbs_region, multi_gpu, tam_block, rept,
                    num_it);
      break;
    default:
      printf("Invalid COALESC");
  }

  return state;
}

// Primeiro Wrapper -- Coalescimento
extern "C" bool ProjectState(float complex* state, int qubits, int proj_qubits,
                             long reg_id, long reg_mask, int multi_gpu) {
  int qbs_coales = 0;
  for (int i = 0; i < qubits; i++) {
    if ((reg_mask >> i) & 1)
      qbs_coales++;
    else
      break;
  }

  int mem_portions = pow(2.0, proj_qubits - qbs_coales);
  int portion_size = 1 << qbs_coales;

  float malloc_size = (1 << proj_qubits) / multi_gpu * sizeof(float complex);
  long inc = ~(reg_mask >> qbs_coales);

  long dev_pos, pos, base = 0;
  for (int d = 0; d < multi_gpu; d++) {
    hipSetDevice(d);
    hipMalloc(&gpu_mem[d], malloc_size);
    error();

    dev_pos = 0;
    for (int b = mem_portions / multi_gpu * d;
         b < mem_portions / multi_gpu * (d + 1); b++) {
      pos = (base << qbs_coales) | reg_id;

      hipMemcpy(gpu_mem[d] + dev_pos, state + pos,
                 portion_size * sizeof(float complex), hipMemcpyHostToDevice);

      base = (base + inc + 1) & ~inc;
      dev_pos += portion_size;
    }
  }

  for (int d = 0; d < multi_gpu; d++) {
    hipSetDevice(d);
    hipDeviceSynchronize();
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_pointer), gpu_mem,
                       multi_gpu * sizeof(hipFloatComplex*));
    error();
  }

  return true;
}

extern "C" bool GetState(float complex* state, int qubits, int proj_qubits,
                         long reg_id, long reg_mask, int multi_gpu) {
  int qbs_coales = 0;
  for (int i = 0; i < qubits; i++) {
    if ((reg_mask >> i) & 1)
      qbs_coales++;
    else
      break;
  }

  int mem_portions = pow(2.0, proj_qubits - qbs_coales);
  int portion_size = 1 << qbs_coales;

  long inc = ~(reg_mask >> qbs_coales);

  long dev_pos, pos, base = 0;
  for (int d = 0; d < multi_gpu; d++) {
    hipSetDevice(d);

    dev_pos = 0;
    for (int b = mem_portions / multi_gpu * d;
         b < mem_portions / multi_gpu * (d + 1); b++) {
      pos = (base << qbs_coales) | reg_id;

      hipMemcpy(state + pos, gpu_mem[d] + dev_pos,
                 portion_size * sizeof(float complex), hipMemcpyDeviceToHost);
      error();
      hipDeviceSynchronize();
      error();

      base = (base + inc + 1) & ~inc;
      dev_pos += portion_size;
    }
  }

  for (int d = 0; d < multi_gpu; d++) {
    hipFree(gpu_mem[d]);
    error();
  }

  return true;
}

bool error() {
  inst++;
  hipError_t e;
  e = hipGetLastError();
  if (e == hipSuccess) return false;
  printf("inst: %d\nerror: %d - %s\n", inst, e, hipGetErrorString(e));
  exit(1);
  return true;
}
