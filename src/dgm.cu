#include "hip/hip_runtime.h"
#include <omp.h>
#include <unistd.h>

#include <cstdio>
#include <iostream>
#include <iterator>

#include "dgm.h"
#include "gpu.h"
#include "pcpu.h"

void Tokenize(const std::string &str, std::vector<std::string> &tokens,
              const std::string &delimiters = ",") {
  // Skip delimiters at beginning.
  std::string::size_type lastPos = str.find_first_not_of(delimiters, 0);
  // Find first "non-delimiter".
  std::string::size_type pos = str.find_first_of(delimiters, lastPos);

  while (std::string::npos != pos || std::string::npos != lastPos) {
    // Found a token, add it to the std::vector.
    tokens.push_back(str.substr(lastPos, pos - lastPos));
    // Skip delimiters.  Note the "not_of"
    lastPos = str.find_first_not_of(delimiters, pos);
    // Find next "non-delimiter"
    pos = str.find_first_of(delimiters, lastPos);
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////

float complex *GenericExecute(float complex *state, std::string function,
                              int qubits, int type, int threads,
                              int factor = 0) {
  DGM dgm;
  dgm.exec_type = type;
  dgm.cpu_params.n_threads = threads;
  dgm.qubits = qubits;
  dgm.factor = factor;

  dgm.setMemory(state);

  dgm.executeFunction(function);

  state = dgm.state;

  dgm.state = NULL;

  return state;
}

float complex *GenericExecute(float complex *state,
                              std::vector<std::string> function, int qubits,
                              int type, int threads, int factor = 0) {
  DGM dgm;
  dgm.exec_type = type;
  dgm.cpu_params.n_threads = threads;
  dgm.qubits = qubits;
  dgm.factor = factor;
  dgm.setMemory(state);

  dgm.executeFunction(function);

  dgm.state = NULL;

  return state;
}

///////////////////////////////////////////////////////////////////////////////////////////////

DGM::DGM() {
  MAX_QB = QB_LIMIT;
  MAX_PT = PT_TAM;

  pts = NULL;
  state = NULL;
  en_print = false;
  exec_type = t_CPU;
  factor = 1;
  gpu_params.multi_gpu = 1;
}

DGM::~DGM() { erase(); }

void DGM::setExecType(int type) { exec_type = type; }

void DGM::printPTs() {
  for (int i = 0; i < vec_pts.size() - 1; i++) {
    vec_pts[i]->print();
  }
}

void DGM::erase() {
  if (!pts) return;

  long i = 0;
  while (pts[i] != NULL) {
    pts[i]->destructor();
    free(pts[i]);
    i++;
  }

  vec_pts.clear();
  pts = NULL;
}

void DGM::allocateMemory() {
  state = (float complex *)calloc(pow(2, qubits), sizeof(float complex));
}

void DGM::setMemory(float complex *mem) {
  freeMemory();
  state = mem;
}

void DGM::freeMemory() {
  if (state) free(state);
  state = NULL;
}

void DGM::setMemoryValue(int pos) { state[pos] = 1; }

int DGM::measure(int q_pos) {
  long size = pow(2.0, qubits);

  long shift = (qubits - 1 - q_pos);

  int count_one, count_zero, num_pb;
  float zero, one, norm, r;
  one = zero = 0;

  // #pragma omp for;
  for (long i = 0; i < size; i++) {
    if ((i >> shift) & 1)
      one += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);
    else
      zero += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);
  }

  long m;
  srand(time(NULL));
  count_one = 0;
  count_zero = 0;
  num_pb = 1;

  for (int i = 0; i < num_pb; i++) {
    r = (double)rand() / RAND_MAX;
    if (zero > r)
      count_zero++;
    else
      count_one++;
  }

  if (count_one > count_zero) {
    measure_value = one;
    norm = sqrt(one);
    m = 1;
  } else {
    measure_value = zero;
    norm = sqrt(zero);
    m = 0;
  }

  long mask;
  mask = pow(2, shift) - 1;
#pragma omp for
  for (long i = 0; i < size / 2; i++) {
    long pos = (i << 1) - (i & mask);
    state[pos] = state[pos | (m << shift)] / norm;
    state[pos | (1 << shift)] = 0.0;
  }

  return m;
}

void DGM::colapse(int q_pos, int value) {
  long size = pow(2.0, qubits);
  long mask = (qubits - 1 - q_pos);

  float m;
  m = 0;

  for (long i = 0; i < size; i++)
    if (((i >> mask) & 1) == value)
      m += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);

  std::cout << m << std::endl;

  m = sqrt(m);
  for (long i = 0; i < size; i++) {
    if (((i >> mask) & 1) == value)
      state[i] = state[i] / m;
    else
      state[i] = 0.0;
  }
}

std::map<long, float> DGM::measure(std::vector<int> q_pos) {
  long mask = 0;

  for (int i = 0; i < q_pos.size(); i++)
    mask = mask | (1 << (qubits - 1 - q_pos[i]));

  std::map<long, float> m;

  long size = pow(2.0, qubits);

  for (long i = 0; i < size; i++)
    m[i & mask] += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);

  return m;
}

void DGM::setFunction(std::string function, int it, bool er) {
  std::vector<std::string> steps;

  Tokenize(function, steps, ";");

  setFunction(steps, it, er);
}

void DGM::setFunction(std::vector<std::string> steps, int it, bool er) {
  if (er)
    erase();
  else
    vec_pts.pop_back();

  std::vector<PT *> step_pts, vec_tmp;
  std::map<long, Group> gps;

  for (long j = 0; j < it; j++)
    for (long i = 0; i < steps.size(); i++) {
      gps = genGroups(steps[i]);
      genPTs(gps, step_pts);

      if (i % 2)
        sort(step_pts.begin(), step_pts.end(), increasing);
      else
        sort(step_pts.begin(), step_pts.end(), decreasing);

      vec_pts.insert(vec_pts.end(), step_pts.begin(), step_pts.end());
    }

  vec_pts.push_back(NULL);

  pts = &vec_pts[0];
}

std::map<long, Group> DGM::genGroups(std::string step) {
  std::vector<std::string> ops;
  Tokenize(step, ops);  // separa os operadores usando "," como delimitador
  qubits = ops.size();

  size_t found_c, found_t, p;
  std::string str;
  long pos, ctrl_value, ctrl_num;

  std::map<long, Group> gps;

  char *pEnd;
  pos = 0;
  std::vector<std::string>::iterator it;
  for (it = ops.begin(); it != ops.end(); ++it) {  // percorre os operadores
    str = *it;
    // std::cout << str << std::endl;
    found_c = str.find("Control");  // tamanho 7
    found_t = str.find("Target");   // tamanho 6
    p = str.find("(") + 1;

    if (found_c != std::string::npos) {  // Controle
      ctrl_num = strtol(str.c_str() + 7, &pEnd, 10);
      ctrl_value = strtol(str.c_str() + p, &pEnd, 10);

      gps[ctrl_num].ctrl.push_back(ctrl_value);  // adicona o valor do controle
      gps[ctrl_num].pos_ctrl.push_back(
          pos);  // e a sua posição ao map relacionado ao controle
    } else if (found_t != std::string::npos) {  // Target
      ctrl_num = strtol(str.c_str() + 6, &pEnd, 10);
      str = str.substr(p, str.size() - p - 1);

      gps[ctrl_num].ops.push_back(str);  // adicona o operador
      gps[ctrl_num].pos_ops.push_back(
          pos);           // e a sua posição ao map relacionado ao target
    } else {              // operador normal
      if (str != "ID") {  // se for ID ignora
        gps[0].ops.push_back(str);      // adiciona o operador
        gps[0].pos_ops.push_back(pos);  // e a sua posição ao map '0'
      }
    }
    pos++;
  }

  return gps;
}

void DGM::genPTs(std::map<long, Group> &gps, std::vector<PT *> &step_pts) {
  step_pts.clear();
  Gates gates;

  std::map<long, Group>::iterator it;
  Group gp;
  PT *pt;
  long ctrl_mask, ctrl_value, ctrl_count;
  long size;

  for (it = gps.begin(); it != gps.end(); ++it) {  // percorre os grupos
    gp = it->second;
    size = gp.ops.size();

    ctrl_count = gp.ctrl.size();
    ctrl_value = ctrl_mask = 0;

    for (long i = 0; i < ctrl_count;
         i++) {  // gera a mascara e o valor do controle (em binario)
      gp.pos_ctrl[i] = qubits - gp.pos_ctrl[i] - 1;
      ctrl_mask += (1 << gp.pos_ctrl[i]);
      if (gp.ctrl[i]) ctrl_value += (1 << gp.pos_ctrl[i]);
    }

    for (int p = 0; p < size; p++) {
      pt = (PT *)malloc(sizeof(PT));
      pt->affected = false;

      pt->qubits = 1;
      pt->start = qubits - gp.pos_ops[p];
      pt->end = pt->start - 1;
      pt->mat_size = 2;

      pt->matrix = gates.getMatrix(gp.ops[p]);

      pt->ctrl_value = ctrl_value;
      pt->ctrl_mask = ctrl_mask;
      pt->ctrl_count = ctrl_count;

      if (ctrl_count) {
        pt->ctrl_pos = (long *)malloc(sizeof(long) * ctrl_count);
        copy(gp.pos_ctrl.begin(), gp.pos_ctrl.end(), pt->ctrl_pos);
      }

      step_pts.push_back(pt);
    }
  }
}

void DGM::genMatrix(float complex *matrix,
                    std::vector<float complex *> &matrices, long tam,
                    long current, long line, long column, float complex cmplx) {
  if (cmplx == 0.0) return;

  if (current == tam) {  // percorreu até a ultima matriz
    matrix[line * (1 << tam) + column] = cmplx;
    return;
  }

  for (long l = 0; l < 2; l++)
    for (long c = 0; c < 2; c++)
      genMatrix(matrix, matrices, tam, current + 1, (line << 1) | l,
                (column << 1) | c, cmplx * matrices[current][l * 2 + c]);
}

void DGM::executeFunction(std::vector<std::string> function, int it) {
  setFunction(function);
  execute(it);
}

void DGM::executeFunction(std::string function, int it) {
  if (function == "") return;

  setFunction(function);
  execute(it);
}

float complex *DGM::execute(int it) {
  float complex *result = state;

  switch (exec_type) {
    case t_CPU:
      CpuExecution1(it);
      break;
    case t_PAR_CPU:
      PCpuExecution1(state, pts, qubits, cpu_params.n_threads,
                     cpu_params.cpu_coales, cpu_params.cpu_region, it);
      break;
    case t_GPU:
      result = GpuExecutionWrapper(state, pts, qubits, gpu_params.gpu_coales,
                                   gpu_params.gpu_region, gpu_params.multi_gpu,
                                   gpu_params.tam_block, gpu_params.rept, it);
      break;
    case t_HYBRID:
      HybridExecution(pts);
      break;
    // case t_DIST:
    //   DistributedExecution(state, pts, qubits, n_threads, cpu_coales,
    //                        cpu_region, it);
    //   break;
    default:
      std::cout << "Erro exec type" << std::endl;
      exit(1);
  }

  return result;
}

OPSCounter DGM::CountOps(int it) {
  OPSCounter counter;

  for (int i = 0; pts[i] != NULL; i++) {
    long mt = pts[i]->matrixType();
    switch (mt) {
      case DENSE:
        (pts[i]->ctrl_mask) ? counter.c_dense++ : counter.dense++;
        break;
      case DIAG_PRI:
        (pts[i]->ctrl_mask) ? counter.c_main_diag++ : counter.main_diag++;
        break;
      case DIAG_SEC:
        (pts[i]->ctrl_mask) ? counter.c_sec_diag++ : counter.sec_diag++;
        break;
      default:
        std::cout << "Error on operator type" << std::endl;
        exit(1);
    }
  }

  counter.dense *= it;
  counter.c_dense *= it;
  counter.main_diag *= it;
  counter.c_main_diag *= it;
  counter.sec_diag *= it;
  counter.c_sec_diag *= it;

  counter.total_op = counter.dense + counter.c_dense + counter.main_diag +
                     counter.c_main_diag + counter.sec_diag +
                     counter.c_sec_diag;
  return counter;
}

void DGM::CpuExecution1(int it) {
  long mem_size = pow(2.0, qubits);

  for (int x = 0; x < it; x++) {
    long i = 0;
    while (pts[i] != NULL) {
      long mt = pts[i]->matrixType();

      switch (mt) {
        case DENSE:
          CpuExecution1_1(pts[i], mem_size);
          break;
        case DIAG_PRI:
          CpuExecution1_2(pts[i], mem_size);
          break;
        case DIAG_SEC:
          CpuExecution1_3(pts[i], mem_size);
          break;
        default:
          exit(1);
      }
      i++;
    }
  }
}

void DGM::CpuExecution1_1(PT *pt, long mem_size) {  // Denso
  long pos0, pos1, shift;

  shift = 1 << pt->end;

  float complex tmp;

  if (!pt->ctrl_count) {  // operador não controlado
    mem_size /= 2;
    for (long pos = 0; pos < mem_size; pos++) {
      pos0 = (pos * 2) - (pos & (shift - 1));
      pos1 = pos0 | shift;

      tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
      state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
      state[pos0] = tmp;
    }
  } else {  // operador controlado
    long mask = ~(pt->ctrl_mask | shift);
    long inc = (~mask) + 1;

    for (long pos = 0; pos < mem_size; pos = (pos + inc) & mask) {
      pos0 = pos | pt->ctrl_value;
      pos1 = pos0 | shift;

      tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
      state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
      state[pos0] = tmp;
    }
  }
}

void DGM::CpuExecution1_2(PT *pt, long mem_size) {  // Diagonal Principal
  long pos0, shift = pt->end;

  if (!pt->ctrl_count)  // operador não controlado
    for (long pos = 0; pos < mem_size; pos++)
      state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];
  else {  // operador controlado
    long mask = ~(pt->ctrl_mask);
    long inc = (~mask) + 1;

    for (long pos = 0; pos < mem_size; pos = (pos + inc) & mask) {
      pos0 = pos | pt->ctrl_value;

      state[pos0] = pt->matrix[((pos0 >> shift) & 1) * 3] * state[pos0];
    }
  }
}

void DGM::CpuExecution1_3(PT *pt, long mem_size) {  // Diagonal Secundária
  long pos0, pos1, shift;

  shift = 1 << pt->end;

  float complex tmp;

  if (!pt->ctrl_count) {  // operador não controlado
    mem_size /= 2;
    for (long pos = 0; pos < mem_size; pos++) {
      pos0 = (pos * 2) - (pos & (shift - 1));
      pos1 = pos0 | shift;

      tmp = pt->matrix[1] * state[pos1];
      state[pos1] = pt->matrix[2] * state[pos0];
      state[pos0] = tmp;
    }
  } else {  // operador controlado
    long mask = ~(pt->ctrl_mask | shift);
    long inc = (~mask) + 1;

    for (long pos = 0; pos < mem_size; pos = (pos + inc) & mask) {
      pos0 = pos | pt->ctrl_value;
      pos1 = pos0 | shift;

      tmp = pt->matrix[1] * state[pos1];
      state[pos1] = pt->matrix[2] * state[pos0];
      state[pos0] = tmp;
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void report_num_threads(int level) {
#pragma omp single
  {
    printf("Level %d: number of threads in the team - %d\n", level,
           omp_get_num_threads());
  }
}

void DGM::HybridExecution(PT **pts) {
  long mem_size = pow(2.0, qubits);
  long qubits_limit = 20;
  long global_coales =
      15;  //(cpu_coales > gpu_coales) ? cpu_coales : gpu_coales;

  long global_region = qubits_limit;
  long global_start, global_end;

  long global_count, global_reg_mask, global_reg_count,
      ext_proj_id;  //, global_pos_count; //atualmente não utilizada

  omp_set_num_threads(cpu_params.n_threads);

  int i = 0;
  while (pts[i] != NULL) {
    global_count = global_coales;
    global_reg_mask = (global_coales) ? (1 << global_coales) - 1 : 0;

    // Realiza a projeção dos operadores de acordo com o limite de qubits que
    // podem ser executados
    global_start = i;
    while (global_count < global_region &&
           pts[i] !=
               NULL) {  // Repete enquanto o número de qubits da região não
                        // atingir o limite (region) e houver operadores
      if (              // pts[i]->matrixType() != DIAG_PRI &&
            // //O qubit de operadores de diagonal principal não importa para
            // região (sempre podem ser acrescentados)
          !((global_reg_mask >> pts[i]->end) & 1)) {
        global_count++;
      }

      if (global_count <=
          global_region)  // && pts[i]->matrixType() != DIAG_PRI)
        global_reg_mask =
            global_reg_mask |
            (1 << pts[i]->end);  // Acrescenta o qubit do operador na região se
                                 // ainda não tiver atingido o limite (region)

      i++;
    }

    while (pts[i] != NULL) {
      if (((global_reg_mask >> pts[i]->end) &
           1))  // || pts[i]->matrixType() == DIAG_PRI)
        i++;
      else
        break;
    }
    global_end = i;

    // Se o número de qubits na região (count) nãoo tiver atingido o limite
    // (region), acrescenta os ultimos qubits (final da mascara) à região até
    // completar for (long a = 1<<(qubits-1); count < region; a = a >> 1){
    for (long a = 1; global_count < global_region; a = a << 1) {
      if (a & ~global_reg_mask) {
        global_reg_mask = global_reg_mask | a;
        global_count++;
      }
    }

    if (global_count < global_region) global_region = global_count;

    global_reg_count = (1 << (qubits - global_region)) +
                       1;  // Número de regiões	- +1 para a condição de parada
                           // incluir todos
    // global_pos_count = 1 << (global_region - 1); // Atualmente não utilizada

    /////////////////////////////////////////////////////////////////////////////////////////////////////

    ext_proj_id = 0;  // contador 'global' do número de regiões já computadas

    // Define a primeira região (reg_id) da thread

#pragma omp parallel num_threads(n_threads)
    {
      if (omp_get_thread_num() != 0) {  // CPU EXECUTION
        long cpu_proj_id;

#pragma omp critical(global_teste)
        {
          cpu_proj_id = ext_proj_id;
          ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
          global_reg_count--;
          if (global_reg_count <= 0) cpu_proj_id = -1;
        }

        while (cpu_proj_id != -1) {
          long cpu_i, cpu_start, cpu_end;

          cpu_start = global_start;

          cpu_i = cpu_start;

          while (cpu_start < global_end) {
            long cpu_count = cpu_params.cpu_coales;
            long cpu_reg_mask =
                (cpu_params.cpu_coales) ? (1 << cpu_params.cpu_coales) - 1 : 0;

            while (
                (cpu_count < cpu_params.cpu_region) &&
                (cpu_i < global_end)) {  // Tem que pertencer a região 'global'
              if (!((cpu_reg_mask >> pts[cpu_i]->end) &
                    1)) {  // Se o qubit do operador estiver fora da região
                           // (reg_mask), incrementa o contador de qubits da
                           // região
                cpu_count++;
              }

              if (cpu_count <= cpu_params.cpu_region)
              // && pts[i]->matrixType() != DIAG_PRI)
              {
                cpu_reg_mask = cpu_reg_mask | (1 << pts[cpu_i]->end);
              }
              // Acrescenta o qubit do operador
              // na região se ainda não tiver
              // atingido o limite (region)

              cpu_i++;
            }

            while (cpu_i < global_end) {
              if (((cpu_reg_mask >> pts[cpu_i]->end) &
                   1))  // || pts[i]->matrixType() == DIAG_PRI)
                cpu_i++;
              else
                break;
            }
            cpu_end = cpu_i;

            for (long a = 1; cpu_count < cpu_params.cpu_region; a = a << 1) {
              if ((a & global_reg_mask) &&
                  (a & ~cpu_reg_mask)) {  // tem que não estar na região da cpu
                                          // e estar na global
                cpu_reg_mask = cpu_reg_mask | a;
                cpu_count++;
              }
            }

            long cpu_reg_count =
                (1 << (global_region - cpu_params.cpu_region)) +
                1;  // Número de regiões 			      -	 +1 para
                    // a condição de parada incluir todos
            long cpu_pos_count = 1 << (cpu_params.cpu_region - 1);
            // Número de posições na região 	-	 -1
            // porque são duas posições por iteração

            long cpu_ext_proj_id = 0;
            long inc_ext_proj_id =
                ~(cpu_reg_mask ^ global_reg_mask) & ((1 << qubits) - 1);

            long proj_id;  // indentificador local da região
            proj_id = cpu_ext_proj_id | cpu_proj_id;
            cpu_ext_proj_id =
                (cpu_ext_proj_id + inc_ext_proj_id + 1) & ~inc_ext_proj_id;
            cpu_reg_count--;

            while (proj_id != -1) {
              // Computa os operadores
              PCpuExecution1_0(state, pts, qubits, cpu_start, cpu_end,
                               cpu_pos_count, proj_id, cpu_reg_mask);

              proj_id = cpu_ext_proj_id | cpu_proj_id;
              cpu_ext_proj_id =
                  (cpu_ext_proj_id + inc_ext_proj_id + 1) & ~inc_ext_proj_id;
              cpu_reg_count--;
              if (cpu_reg_count <= 0) proj_id = -1;
            }

            cpu_start = cpu_end;
          }

#pragma omp critical(global_teste)
          {
            cpu_proj_id = ext_proj_id;
            ext_proj_id =
                (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
            global_reg_count--;
            if (global_reg_count <= 0) cpu_proj_id = -1;
          }
        }

      }
      // #pragma omp section          //GPU EXECUTION
      else {
        long gpu_proj_id;

#pragma omp critical(global_teste)
        {
          gpu_proj_id = ext_proj_id;
          ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
          global_reg_count--;
          if (global_reg_count <= 0) gpu_proj_id = -1;
        }

        while (gpu_proj_id != -1) {
          // Project Gates
          std::vector<PT *> gpu_pts;

          int gpu_i;

          int map_qb[qubits];
          memset(map_qb, -1, qubits * sizeof(int));

          int m = 0;
          for (gpu_i = 0; gpu_i < qubits; gpu_i++) {
            if ((1 << gpu_i) & global_reg_mask) {
              map_qb[gpu_i] = m++;
            }
          }

          PT *aux;
          gpu_pts.clear();
          for (int gpu_i = global_start; gpu_i < global_end; gpu_i++) {
            // verifica se o controle do operador satisfaz a parte global da
            // região
            if ((pts[gpu_i]->ctrl_mask & gpu_proj_id & ~global_reg_mask) ==
                (pts[gpu_i]->ctrl_value & ~global_reg_mask)) {
              aux = new PT();

              aux->qubits = pts[gpu_i]->qubits;

              aux->matrix = pts[gpu_i]->matrix;
              aux->mat_size = pts[gpu_i]->mat_size;
              aux->ctrl_mask = pts[gpu_i]->ctrl_mask & global_reg_mask;
              aux->ctrl_value = pts[gpu_i]->ctrl_value & global_reg_mask;

              aux->end = map_qb[pts[gpu_i]->end];
              aux->start = aux->end - log2((float)aux->mat_size);

              aux->ctrl_count = 0;
              for (int c = global_coales; c < qubits; c++) {
                if (aux->ctrl_mask & (1 << c)) {
                  aux->ctrl_count++;

                  aux->ctrl_mask &= ~(
                      1
                      << c);  // retira da mascara o controle do qubit atual (c)
                  aux->ctrl_mask |= (1 << map_qb[c]);  // e coloca o qubit que
                                                       // ele mapeia (map_qb[c])

                  if (aux->ctrl_value &
                      (1 << c)) {  // se o valor do controle for zero faz a
                                   // mesma coisa para ctrl_value;
                    aux->ctrl_mask &= ~(1 << c);
                    aux->ctrl_mask |= (1 << map_qb[c]);
                  }
                }
              }

              gpu_pts.push_back(aux);
            }
          }
          gpu_pts.push_back(NULL);
          ////////////////

          ProjectState(state, qubits, global_region, gpu_proj_id,
                       global_reg_mask, gpu_params.multi_gpu);

          GpuExecutionWrapper(NULL, &gpu_pts[0], global_region,
                              gpu_params.gpu_coales, gpu_params.gpu_region,
                              gpu_params.multi_gpu, gpu_params.tam_block,
                              gpu_params.rept, 1);

          GetState(state, qubits, global_region, gpu_proj_id, global_reg_mask,
                   gpu_params.multi_gpu);

          for (int c = 0; c < gpu_pts.size() - 1; c++) {
            delete gpu_pts[c];
          }

#pragma omp critical(global_teste)
          {
            gpu_proj_id = ext_proj_id;
            ext_proj_id =
                (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
            global_reg_count--;
            if (global_reg_count <= 0) gpu_proj_id = -1;
          }
        }
      }
      //}
    }
  }
}

void DGM::setCpuStructure(long cpu_region, long cpu_coales) {
  this->cpu_params.cpu_region = cpu_region;
  this->cpu_params.cpu_coales = cpu_coales;
}

void DGM::setGpuStructure(long gpu_region, long gpu_coales, int rept) {
  this->gpu_params.gpu_region = gpu_region;
  this->gpu_params.gpu_coales = gpu_coales;
  this->gpu_params.rept = rept;
  this->gpu_params.tam_block = 1 << gpu_region / 2 / rept;
}

// Coalescimento
void MPI_coalesc(float complex *state, int qubits, int proj_qubits, long reg_id,
                 long reg_mask, int world_size) {
  int qbs_coales = 0;
  for (int i = 0; i < qubits; i++) {
    if ((reg_mask >> i) & 1) {
      qbs_coales++;
    } else {
      break;
    }
  }

  int mem_portions = pow(2.0, proj_qubits - qbs_coales);
  int portion_size = 1 << qbs_coales;

  float malloc_size = (1 << proj_qubits) * sizeof(float complex);

  float complex *new_state =
      (float complex *)(malloc(sizeof(float complex) * pow(2, qubits)));
  int *chunk_sizes = (int *)(malloc(sizeof(int) * world_size));
  int *displ = (int *)(malloc(sizeof(int) * world_size));

  long inc = ~(reg_mask >> qbs_coales);

  long dest_pos, src_pos, base = 0;
  for (int d = 0; d < world_size; d++) {
    displ[d] = dest_pos;
    for (int b = mem_portions / world_size * d;
         b < mem_portions / world_size * (d + 1); b++) {
      src_pos = (base << qbs_coales) | reg_id;

      memcpy(new_state + dest_pos, state + src_pos,
             portion_size * sizeof(float complex));

      base = (base + inc + 1) & ~inc;
      dest_pos += portion_size;
    }
    chunk_sizes[d] = dest_pos - displ[d];
  }
}

// void DistributedExecution(float complex *state, PT **pts, int qubits,
//                           long n_threads, int coales, int region, int it) {}