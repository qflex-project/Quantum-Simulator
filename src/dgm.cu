#include "hip/hip_runtime.h"
#include <iostream>
#include "dgm.h"
#include <omp.h>
#include <unistd.h>
#include <cstdio>
#include <iterator>

void Tokenize(const string& str, vector<string>& tokens, const string& delimiters = ",")
{
	// Skip delimiters at beginning.
	string::size_type lastPos = str.find_first_not_of(delimiters, 0);
	// Find first "non-delimiter".
	string::size_type pos = str.find_first_of(delimiters, lastPos);

	while (string::npos != pos || string::npos != lastPos)
	{
		// Found a token, add it to the vector.
		tokens.push_back(str.substr(lastPos, pos - lastPos));
		// Skip delimiters.  Note the "not_of"
		lastPos = str.find_first_not_of(delimiters, pos);
		// Find next "non-delimiter"
		pos = str.find_first_of(delimiters, lastPos);
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////

float complex* GenericExecute(float complex *state, string function, int qubits, int type, int threads, int factor = 0){
	DGM dgm;
	dgm.exec_type = type;
	dgm.n_threads = threads;
	dgm.qubits = qubits;
	dgm.factor = factor;

	dgm.setMemory(state);

	dgm.executeFunction(function);

	state = dgm.state;

	dgm.state = NULL;

	return state;
}

float complex* GenericExecute(float complex *state, vector<string> function, int qubits, int type, int threads, int factor = 0){
	DGM dgm;
	dgm.exec_type = type;
	dgm.n_threads = threads;
	dgm.qubits = qubits;
	dgm.factor = factor;
	dgm.setMemory(state);

	dgm.executeFunction(function);

	dgm.state = NULL;

	return state;
}

///////////////////////////////////////////////////////////////////////////////////////////////

DGM::DGM(){
	MAX_QB = QB_LIMIT;
	MAX_PT = PT_TAM;

	pts = NULL;
	state = NULL;
	en_print = false;
	exec_type = t_CPU;
	factor = 1;
	multi_gpu = 1;
}

DGM::~DGM(){erase();}

void DGM::setExecType(int type){
	exec_type = type;
}

void DGM::printPTs(){
	for (int i = 0; i < vec_pts.size() -1; i++){
		vec_pts[i]->print();
	}
}

void DGM::erase(){
	if (!pts) return;

	long i = 0;
	while (pts[i] != NULL){
		pts[i]->destructor();
		free(pts[i]);
		i++;
	}

	vec_pts.clear();
	pts = NULL;
}

void DGM::allocateMemory(){
	state = (float complex*) calloc(pow(2, qubits), sizeof(float complex));
}

void DGM::setMemory(float complex* mem){
	freeMemory();
	state = mem;
}

void DGM::freeMemory(){
	if (state) free(state);
	state = NULL;
}

void DGM::setMemoryValue(int pos){
	state[pos] = 1;
}

int DGM::measure(int q_pos){
	long size = pow(2.0, qubits);

	long shift = (qubits - 1 - q_pos);

	int count_one, count_zero, num_pb;
	float zero, one, norm, r;
	one = zero = 0;

	//#pragma omp for;
	for (long i = 0; i < size; i++){
		if ((i >> shift) & 1)
			one += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);
		else
			zero += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);
	}

	long m;
	srand (time(NULL));
	count_one = 0;
	count_zero = 0;	
	num_pb = 1;

	for (int i = 0; i < num_pb; i++){
		r = (double) rand() / RAND_MAX;
		if (zero > r) count_zero++;
		else count_one++;
	}

	if (count_one > count_zero){
		measure_value = one;
		norm = sqrt(one);
		m = 1;
	}
	else{
		measure_value = zero;
		norm = sqrt(zero);
		m = 0;
	}

	long mask;
	mask = pow(2, shift) - 1;
	#pragma omp for
	for (long i = 0; i < size/2; i++){
		long pos = (i << 1) - (i&mask);
		state[pos] = state[pos | (m << shift)]/norm;
		state[pos | (1<<shift)] = 0.0;
	}

	return m;
}

void DGM::colapse(int q_pos, int value){
	long size = pow(2.0, qubits);
	long mask = (qubits - 1 - q_pos);

	float m;
	m = 0;

	for (long i = 0; i < size; i++)
		if (((i >> mask)&1) == value) m += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);

	cout << m << endl;

	m = sqrt(m);
	for (long i = 0; i < size; i++){
		if (((i >> mask)&1) == value) state[i] = state[i]/m;
		else state[i] = 0.0;
	}
}

map <long, float> DGM::measure(vector<int> q_pos){
	long mask = 0;

	for (int i =0; i < q_pos.size(); i++) mask = mask | (1<<(qubits - 1 - q_pos[i]));

	map <long, float> m;

	long size = pow(2.0, qubits);

	for (long i =0; i < size; i++) m[i&mask] += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);

	return m;
}

void DGM::setFunction(string function, int it, bool er){
	vector <string> steps;

	Tokenize(function, steps, ";");

	setFunction(steps, it, er);
}

void DGM::setFunction(vector <string> steps, int it, bool er){
	if (er) erase();
	else vec_pts.pop_back();


	vector <PT*> step_pts, vec_tmp;
	map<long, Group> gps;

	for (long j = 0; j< it; j++)
	for (long i = 0; i < steps.size(); i++){
		gps = genGroups(steps[i]);
		genPTs(gps, step_pts);

		if (i%2)
			sort(step_pts.begin(), step_pts.end(), increasing);
		else
			sort(step_pts.begin(), step_pts.end(), decreasing);

		vec_pts.insert(vec_pts.end(), step_pts.begin(), step_pts.end());
	}

	vec_pts.push_back(NULL);

	pts = &vec_pts[0];
}

map <long, Group> DGM::genGroups(string step){
	vector <string> ops;
	Tokenize(step, ops); //separa os operadores usando "," como delimitador
	qubits = ops.size();

	size_t found_c, found_t, p;
	string str;
	long pos, ctrl_value, ctrl_num;
	
	map<long, Group> gps;

	char * pEnd;
	pos = 0;
	vector<string>::iterator it;
	for (it = ops.begin() ; it != ops.end(); ++it){ //percorre os operadores
		str = *it;
		//cout << str << endl;
		found_c = str.find("Control"); //tamanho 7
		found_t = str.find("Target");  //tamanho 6
		p = str.find("(") + 1;

		if (found_c != string::npos){ //Controle
			ctrl_num = strtol(str.c_str()+7, &pEnd, 10);
			ctrl_value = strtol(str.c_str()+p, &pEnd, 10);

			gps[ctrl_num].ctrl.push_back(ctrl_value); //adicona o valor do controle
			gps[ctrl_num].pos_ctrl.push_back(pos);  //e a sua posição ao map relacionado ao controle
		}
		else if(found_t != string::npos){ //Target
			ctrl_num = strtol(str.c_str()+6, &pEnd, 10);
			str = str.substr(p, str.size()-p-1);

			gps[ctrl_num].ops.push_back(str);     //adicona o operador
			gps[ctrl_num].pos_ops.push_back(pos); //e a sua posição ao map relacionado ao target
		}
		else{ //operador normal
			if (str != "ID"){ //se for ID ignora
				gps[0].ops.push_back(str);     //adiciona o operador
				gps[0].pos_ops.push_back(pos); //e a sua posição ao map '0'
			}
		}
		pos++;
	}
	
	return gps;
}

void DGM::genPTs(map<long, Group> &gps, vector <PT*> &step_pts){
	step_pts.clear();
	Gates gates;

	map<long,Group>::iterator it;	
	Group gp;
	PT* pt;
	long ctrl_mask, ctrl_value, ctrl_count;
	long size;
	
	for (it = gps.begin(); it != gps.end(); ++it){ //percorre os grupos
		gp = it->second;
		size = gp.ops.size();
		
		ctrl_count = gp.ctrl.size();
		ctrl_value = ctrl_mask = 0;

		for (long i = 0; i < ctrl_count; i++){ //gera a mascara e o valor do controle (em binario)
			gp.pos_ctrl[i] =  qubits - gp.pos_ctrl[i] - 1;
			ctrl_mask += (1 << gp.pos_ctrl[i]);
			if (gp.ctrl[i]) ctrl_value += (1 << gp.pos_ctrl[i]);
		}

		for (int p = 0; p < size; p++){
			
			pt = (PT*) malloc(sizeof(PT));
			pt->affected = false;

			pt->qubits = 1;
			pt->start = qubits - gp.pos_ops[p];
			pt->end = pt->start - 1;
			pt->mat_size = 2;
			
			pt->matrix = gates.getMatrix(gp.ops[p]);

			pt->ctrl_value = ctrl_value;
			pt->ctrl_mask = ctrl_mask;
			pt->ctrl_count = ctrl_count;

			if (ctrl_count){
				pt->ctrl_pos = (long*)malloc(sizeof(long) * ctrl_count);
				copy(gp.pos_ctrl.begin(), gp.pos_ctrl.end(), pt->ctrl_pos);
			}

			step_pts.push_back(pt);
		}
	}
}

void DGM::genMatrix(float complex* matrix, vector<float complex*> &matrices, long tam, long current, long line, long column, float complex cmplx){
	if (cmplx == 0.0) return;

	if (current == tam){ //percorreu até a ultima matriz
		matrix[line*(1<<tam) + column] = cmplx;
		return;
	}

	for (long l = 0; l < 2; l++)
		for (long c = 0; c < 2; c++)
			genMatrix(matrix, matrices, tam, current+1, (line<<1)|l, (column<<1)|c, cmplx * matrices[current][l*2+c]);
}


void DGM::executeFunction(vector <string> function, int it){
	setFunction(function);
	execute(it);
}

void DGM::executeFunction(string function, int it){
	if (function == "") return;

	setFunction(function);
	execute(it);
}


float complex* DGM::execute(int it){
	float complex* result = state;

	switch (exec_type){
		case t_CPU:
			CpuExecution1(it);
			break;
		case t_PAR_CPU:
			PCpuExecution1(state, pts, qubits, n_threads, cpu_coales, cpu_region, it);
			break;
		case t_GPU:
			result = GpuExecutionWrapper(state, pts, qubits, gpu_coales, gpu_region, multi_gpu, tam_block, rept, it);
			break;
		case t_HYBRID:
			HybridExecution(pts);
			break;
		default:
			cout << "Erro exec type" << endl;
			exit(1);
	}

	return result;
}


void DGM::CountOps(int it){
	dense = main_diag = sec_diag = c_dense = c_main_diag = c_sec_diag = 0;

	for (int i =0; pts[i]!=NULL; i++){
		long mt = pts[i]->matrixType();
		switch (mt){
			case DENSE:
				(pts[i]->ctrl_mask) ? c_dense++ : dense++;
				break;
			case DIAG_PRI:
				(pts[i]->ctrl_mask) ? c_main_diag++ : main_diag++;
				break;
			case DIAG_SEC:
				(pts[i]->ctrl_mask) ? c_sec_diag++ : sec_diag++;
				break;
			default:
				cout << "Error on operator type" << endl;
				exit(1);
		}
	}

	dense *= it;
	c_dense *= it;
	main_diag *= it;
	c_main_diag *= it;
	sec_diag *= it;
	c_sec_diag *= it;

	total_op = dense + c_dense + main_diag + c_main_diag + sec_diag + c_sec_diag;
}

void DGM::CpuExecution1(int it){
	long mem_size = pow(2.0, qubits);

	for (int x = 0; x < it; x++){
		long i = 0;
		while (pts[i] != NULL){
			long mt = pts[i]->matrixType();

			switch (mt){
				case DENSE:
					CpuExecution1_1(pts[i], mem_size);
					break;
				case DIAG_PRI:
					CpuExecution1_2(pts[i], mem_size);
					break;
				case DIAG_SEC:
					CpuExecution1_3(pts[i], mem_size);
					break;
				default:
					exit(1);
			}
			i++;
		}
	}
}

void DGM::CpuExecution1_1(PT *pt, long mem_size){ //Denso
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;
	
	float complex tmp;
		
	if (!pt->ctrl_count){ 			//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		long mask = ~(pt->ctrl_mask | shift);
		long inc = (~mask) + 1;

		for (long pos = 0; pos < mem_size; pos = (pos+inc) & mask){
			pos0 = pos | pt->ctrl_value;
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];			
			state[pos0] = tmp;
		}
	}
}

void DGM::CpuExecution1_2(PT *pt, long mem_size){ //Diagonal Principal
	long pos0, shift = pt->end;
		
	if (!pt->ctrl_count)	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++)
			state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];
	else{					//operador controlado
		long mask = ~(pt->ctrl_mask);
		long inc = (~mask) + 1;

		for (long pos = 0; pos < mem_size; pos = (pos+inc) & mask){
			pos0 = pos | pt->ctrl_value;

			state[pos0] = pt->matrix[((pos0 >> shift) & 1) * 3] * state[pos0];
		}
	}
}

void DGM::CpuExecution1_3(PT *pt, long mem_size){ //Diagonal Secundária
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;

	float complex tmp;
		
	if (!pt->ctrl_count){ 	//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		long mask = ~(pt->ctrl_mask | shift);
		long inc = (~mask) + 1;
		
		for (long pos = 0; pos < mem_size; pos = (pos+inc) & mask){
			pos0 = pos | pt->ctrl_value;
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void DGM::CpuExecution2_1(PT *pt, long mem_size){ //Denso
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;
	mem_size /= 2;

	float complex tmp;
		
	if (!pt->ctrl_count) 			//operador não controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
			state[pos0] = tmp;
		}
	else{					//operador controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;
			if ((pos0 & pt->ctrl_mask) == pt->ctrl_value){
				tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
				state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];			
				state[pos0] = tmp;
			}
		}
		cout << endl;
	}
}

void DGM::CpuExecution2_2(PT *pt, long mem_size){ //Diagonal Principal
	long shift = pt->end;
		
	if (!pt->ctrl_count)	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++)
			state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];
	else					//operador controlado
		for (long pos = 0; pos < mem_size; pos++)
			if ((pos & pt->ctrl_mask) == pt->ctrl_value)
				state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];

}



void DGM::CpuExecution2_3(PT *pt, long mem_size){ //Diagonal Secundária
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;
	mem_size /= 2;

	float complex tmp;
		
	if (!pt->ctrl_count) 	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	else					//operador controlado
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;
			if ((pos0 & pt->ctrl_mask) == pt->ctrl_value){
				tmp = pt->matrix[1] * state[pos1];
				state[pos1] = pt->matrix[2] * state[pos0];
				state[pos0] = tmp;
			}
		}
	
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void DGM::CpuExecution3_1(PT *pt, long mem_size){ //Denso
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;

	float complex tmp;
		
	if (!pt->ctrl_count){ 			//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		vector <long> gap, max;
		long i, c, mask;

		mask = pt->ctrl_mask | shift;

		c = 0;
		for (i = 0; i < qubits; i++){
			if (((mask >> i) & 1) == 0) c++;
			else if (c){
				gap.push_back(1<<(i-c));
				max.push_back(1<<i);
				c = 0;
			}
		}
		if (c){
			gap.push_back(1<<(i-c));
			max.push_back(1<<(qubits+1));
		}
		else{	
			gap.push_back(1<<(qubits+1));
			max.push_back(1<<(qubits+2));
		}

		long pos = 0;

		while (pos < mem_size){
				pos0 = pos | pt->ctrl_value;
				pos1 = pos0 | shift;

				//cout << pos0 <<  " " << pos1 << endl; 

				tmp = pt->matrix[0] * state[pos0] + pt->matrix[1] * state[pos1];
				state[pos1] = pt->matrix[2] * state[pos0] + pt->matrix[3] * state[pos1];			
				state[pos0] = tmp;

				pos += gap[0];
				i = 0;
				while (pos & max[i]){
					pos ^= max[i++];
					pos += gap[i];
				}

		}
		//cout << endl;
	}	
}

void DGM::CpuExecution3_2(PT *pt, long mem_size){ //Diagonal Principal
	long pos0, shift = pt->end;
		
	if (!pt->ctrl_count)	//operador não controlado
		for (long pos = 0; pos < mem_size; pos++)
			state[pos] = pt->matrix[((pos >> shift) & 1) * 3] * state[pos];
	else{					//operador controlado
		vector <long> gap, max;
		long i, c, mask;

		mask = pt->ctrl_mask;

		c = 0;
		for (i = 0; i < qubits; i++){
			if (((mask >> i) & 1) == 0) c++;
			else if (c){
				gap.push_back(1<<(i-c));
				max.push_back(1<<i);
				c = 0;
			}
		}
		if (c){
			gap.push_back(1<<(i-c));
			max.push_back(1<<(qubits+1));
		}
		else{	
			gap.push_back(1<<(qubits+1));
			max.push_back(1<<(qubits+2));
		}

		long pos = 0;

		while (pos < mem_size){
				pos0 = pos | pt->ctrl_value;

				//cout << pos0 << endl; 
				state[pos0] = pt->matrix[((pos0 >> shift) & 1) * 3] * state[pos0];

				pos += gap[0];
				i = 0;
				while (pos & max[i]){
					pos ^= max[i++];
					pos += gap[i];
				}

		}
	}
}

void DGM::CpuExecution3_3(PT *pt, long mem_size){ //Diagonal Secundária
	long pos0, pos1, shift;
	
	shift = 1 << pt->end;


	float complex tmp;
		
	if (!pt->ctrl_count){ 	//operador não controlado
		mem_size /= 2;
		for (long pos = 0; pos < mem_size; pos++){
			pos0 = (pos * 2) - (pos & (shift-1));
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;
		}
	}
	else{					//operador controlado
		vector <long> gap, max;
		long i, c, mask;

		mask = pt->ctrl_mask | shift;

		c = 0;
		for (i = 0; i < qubits; i++){
			if (((mask >> i) & 1) == 0) c++;
			else if (c){
				gap.push_back(1<<(i-c));
				max.push_back(1<<i);
				c = 0;
			}
		}
		if (c){
			gap.push_back(1<<(i-c));
			max.push_back(1<<(qubits+1));
		}
		else{	
			gap.push_back(1<<(qubits+1));
			max.push_back(1<<(qubits+2));
		}

		long pos = 0;

		while (pos < mem_size){
			pos0 = pos | pt->ctrl_value;
			pos1 = pos0 | shift;

			tmp = pt->matrix[1] * state[pos1];
			state[pos1] = pt->matrix[2] * state[pos0];
			state[pos0] = tmp;

			pos += gap[0];
			i = 0;
			while (pos & max[i]){
				pos ^= max[i++];
				pos += gap[i];
			}
		}
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void PCpuExecution1(float complex *state, PT **pts, int qubits, long n_threads, int coales, int region, int it){
	long i, start, end;
	i = start = 0;

	long max_reg_count = (1 << (qubits - region));
	long* reg_ids = (long*) malloc((max_reg_count)*sizeof(long));

	while (pts[i] != NULL){
		long count = coales;
		long reg_mask = (coales)? (1 << coales) - 1 : 0;

		//Pega os operadores que estão dentro da região coalescida (reg_mask inicial),
		//e acrescenta operadores em qubits fora dela até chegar ao limite da região (region definida)
		start = i;
		while (count < region && pts[i] != NULL){					//Repete enquanto o número de qubits da região não atingir o limite (region) e houver operadores
			if (//pts[i]->matrixType() != DIAG_PRI &&					//O qubit de operadores de diagonal principal não importa para região (sempre podem ser acrescentados)
				!((reg_mask >> pts[i]->end) & 1)){				//Se o qubit do operador estiver fora da região (reg_mask), incrementa o contador de qubits da região
				count++;
			}

			if (count <= region)// && pts[i]->matrixType() != DIAG_PRI)
				reg_mask = reg_mask | (1 << pts[i]->end);			//Acrescenta o qubit do operador na região se ainda não tiver atingido o limite (region)
				
			i++;
		}
		//Segue acerscentado até encontrar um operador que não esteja dentro da região
		while (pts[i] != NULL){
			if (((reg_mask >> pts[i]->end) & 1))// || pts[i]->matrixType() == DIAG_PRI)
				i++;
			else
				break;
		}
		end = i;	//Executa até o operador na posiçao 'i' (exclusive) nesta iteração


		//Se o número de qubits na região (count) não tiver atingido o limite (region),
		//acrescenta os ultimos qubits (final da mascara) à região até completar
		//for (long a = 1<<(qubits-1); count < region; a = a >> 1){
		for (long a = 1; count < region; a = a << 1){
			if (a & ~reg_mask){
				reg_mask = reg_mask | a;
				count++;
			}
		}

		if (count < region)
			region = count;

		long reg_count = (1 << (qubits - region));	//Número de regiões
		long pos_count = 1 << (region - 1);			//Número de posições na região: -1 porque são duas posições por iteração

		omp_set_num_threads(n_threads);

		long ext_reg_id = 0;	//contador 'global' do número de regiões já computadas

		for (size_t j = 0; j < reg_count; j++)	
		{
			reg_ids[j] = ext_reg_id;
			ext_reg_id = (ext_reg_id + reg_mask + 1) & ~reg_mask;
		}

		#pragma omp parallel for schedule(runtime)
		for (size_t j = 0; j < reg_count; j++) {
			PCpuExecution1_0(state, pts, qubits, start, end, pos_count, reg_ids[j], reg_mask);
		}

	}
	free(reg_ids);
}

void PCpuExecution1_0(float complex *state, PT **pts, int qubits, int start, int end, int pos_count, int reg_id, int reg_mask){
	PT *QG;
	long pos0, pos1;
	float complex tmp;


	for (int op = start; op < end; op++){
		QG = pts[op];
		long shift = (1 << QG->end);	//mascara com a posição do qubit do operador
		long mt = QG->matrixType();
		//if (mt == DIAG_PRI) shift = coalesc;	//se for um operador de diagonal principal, a posição do qubit não é relevante
		long pos_mask = reg_mask & ~shift;	//mascara da posição --- retira o 'shift' da reg_mask, para o 'inc pular sobre ' esse bit também
		long inc = ~pos_mask + 1;	//usado para calcular a proxima posição de uma região
		long pos = 0;
					
		if (!QG->ctrl_count){
			switch (mt){
				case DENSE:
					for (long p = 0; p < pos_count; p++){
						pos0 = pos | reg_id;
						pos1 = pos0 | shift;
						pos = (pos+inc) & pos_mask;

						tmp 		= QG->matrix[2] * state[pos0] + QG->matrix[3] * state[pos1];
						state[pos0] = QG->matrix[0] * state[pos0] + QG->matrix[1] * state[pos1];
						state[pos1] = tmp;
					}
					break;
				case DIAG_PRI:
					for (long p = 0; p < pos_count; p++){
							pos0 = pos | reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp			= QG->matrix[3] * state[pos1];
							state[pos0] *= QG->matrix[0];// * state[pos0];
							state[pos1] = tmp;// * state[pos1];tmp;
					}
					break;
				
				case DIAG_SEC:
					for (long p = 0; p < pos_count; p++){
							pos0 = pos | reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp 		= QG->matrix[2] * state[pos0];
							state[pos0] = QG->matrix[1] * state[pos1];
							state[pos1] = tmp;
					}
					break;
				default:
					printf("Erro de Tipo\n");
			}
		}
		//Importante: reg_id é o identificador da região e corresponde ao valor dos qubits externos à região de operação (reg_mask)
		else {			
			if ((QG->ctrl_mask & reg_id & ~reg_mask) == (QG->ctrl_value & ~reg_mask)){		//Verifica se a parte 'global' do controle satisfaz a região (reg_id)

				// É preciso arrumar o reg_mask retirando os qubits de controle que estão dentro da região e arrumar o reg_id para incluir o valor dos controles
				long ctrl_reg_id = reg_id | QG->ctrl_value;				//Esta operação inclui o valor dos controles locais no reg_id (funciona pois os valores globais já deram match)
				long ctrl_reg_mask = reg_mask;							//Valor inicial da mascara da região com controle
				long ctrl_pos_count = pos_count;						//Número inicial de posições a serem calculadas

				for (int i = 0, m = 1; i < qubits; i++, m = m << 1){ 	//percorre os qubits
					if (m & reg_mask & QG->ctrl_mask){					//se o qubit pertencer a região e for um controle:
						ctrl_reg_mask ^= m;								//	remove ele da região(reg_mask) (para não iterar sobre ele)
						ctrl_pos_count /= 2;							//	diminui a quantidade de posições que é preciso calcular.
					}
				}

				pos_mask = ctrl_reg_mask & ~shift;						//mascara da posição --- retira o 'shift' da reg_mask, para o 'inc pular sobre' esse bit também
				inc = ~pos_mask + 1;

				switch (mt){
					case DENSE:
						for (long p = 0; p < ctrl_pos_count; p++){
							pos0 = pos | ctrl_reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp 		= QG->matrix[2] * state[pos0] + QG->matrix[3] * state[pos1];
							state[pos0] = QG->matrix[0] * state[pos0] + QG->matrix[1] * state[pos1];
							state[pos1] = tmp;
						}
						break;
					case DIAG_PRI:
						for (long p = 0; p < ctrl_pos_count; p++){
							pos0 = pos | ctrl_reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp			= QG->matrix[3] * state[pos1];
							state[pos0] *= QG->matrix[0];
							state[pos1] = tmp;
						}
						break;
					
					case DIAG_SEC:
						for (long p = 0; p < ctrl_pos_count; p++){
							pos0 = pos | ctrl_reg_id;
							pos1 = pos0 | shift;
							pos = (pos+inc) & pos_mask;

							tmp 		= QG->matrix[2] * state[pos0];
							state[pos0] = QG->matrix[1] * state[pos1];
							state[pos1] = tmp;
						}
						break;

					default:
						printf("Erro de Tipo");
				}
			}
		}
	}
}

void report_num_threads(int level){
	#pragma omp single
	{
		printf("Level %d: number of threads in the team - %d\n", level, omp_get_num_threads());
	}
}

void DGM::HybridExecution(PT **pts){
	long mem_size = pow(2.0, qubits);
	long qubits_limit = 20;
	long global_coales = 15; //(cpu_coales > gpu_coales) ? cpu_coales : gpu_coales;

	long global_region = qubits_limit;
	long global_start, global_end;

	long global_count, global_reg_mask, global_reg_count, ext_proj_id; //, global_pos_count; //atualmente não utilizada

	omp_set_num_threads(n_threads);

	int i = 0;
	while (pts[i] != NULL){
		global_count = global_coales;
		global_reg_mask = (global_coales)? (1 << global_coales) - 1 : 0;

		//Realiza a projeção dos operadores de acordo com o limite de qubits que podem ser executados
		global_start = i;
		while (global_count < global_region && pts[i] != NULL){			//Repete enquanto o número de qubits da região não atingir o limite (region) e houver operadores
			if (//pts[i]->matrixType() != DIAG_PRI &&					//O qubit de operadores de diagonal principal não importa para região (sempre podem ser acrescentados)
			!((global_reg_mask >> pts[i]->end) & 1)){				
				global_count++;
			}

			if (global_count <= global_region)// && pts[i]->matrixType() != DIAG_PRI)
				global_reg_mask = global_reg_mask | (1 << pts[i]->end);			//Acrescenta o qubit do operador na região se ainda não tiver atingido o limite (region)	

			i++;
		}

		while (pts[i] != NULL){
			if (((global_reg_mask >> pts[i]->end) & 1))// || pts[i]->matrixType() == DIAG_PRI)
				i++;
			else
				break;
		}
		global_end = i;

		//Se o número de qubits na região (count) nãoo tiver atingido o limite (region),
		//acrescenta os ultimos qubits (final da mascara) à região até completar
		//for (long a = 1<<(qubits-1); count < region; a = a >> 1){
		for (long a = 1; global_count < global_region; a = a << 1){
			if (a & ~global_reg_mask){
				global_reg_mask = global_reg_mask | a;
				global_count++;
			}
		}

		if (global_count < global_region)
			global_region = global_count;
	
		global_reg_count = (1 << (qubits - global_region)) + 1; 				//Número de regiões	- +1 para a condição de parada incluir todos
		// global_pos_count = 1 << (global_region - 1); // Atualmente não utilizada

		/////////////////////////////////////////////////////////////////////////////////////////////////////

		ext_proj_id = 0;	//contador 'global' do número de regiões já computadas

		//Define a primeira região (reg_id) da thread

		#pragma omp parallel num_threads(n_threads)
		{
			if (omp_get_thread_num()!=0){  //CPU EXECUTION
				long cpu_proj_id;		
				
				#pragma omp critical (global_teste)
				{
					cpu_proj_id = ext_proj_id;
					ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
					global_reg_count--;
					if (global_reg_count <= 0)
						cpu_proj_id = -1;
				}
	
				while (cpu_proj_id != -1){
					long cpu_i, cpu_start, cpu_end;

					cpu_start = global_start;
			
					cpu_i = cpu_start;
			
					while (cpu_start < global_end){
						long cpu_count = cpu_coales;
						long cpu_reg_mask = (cpu_coales)? (1 << cpu_coales) - 1 : 0;
			
						while ((cpu_count < cpu_region) && (cpu_i < global_end)){	//Tem que pertencer a região 'global'
							if (!((cpu_reg_mask >> pts[cpu_i]->end) & 1)){			//Se o qubit do operador estiver fora da região (reg_mask), incrementa o contador de qubits da região
								cpu_count++;
							}
		
							if (cpu_count <= cpu_region)// && pts[i]->matrixType() != DIAG_PRI)
								cpu_reg_mask = cpu_reg_mask | (1 << pts[cpu_i]->end);	//Acrescenta o qubit do operador na região se ainda não tiver atingido o limite (region)
						
							cpu_i++;
						}
			
						while (cpu_i < global_end){
							if (((cpu_reg_mask >> pts[cpu_i]->end) & 1))// || pts[i]->matrixType() == DIAG_PRI)
								cpu_i++;
							else
								break;
						}
						cpu_end = cpu_i;
			
						for (long a = 1; cpu_count < cpu_region; a = a << 1){
							if ((a & global_reg_mask) && (a & ~cpu_reg_mask)){ //tem que não estar na região da cpu e estar na global
								cpu_reg_mask = cpu_reg_mask | a;
								cpu_count++;
							}
						}
	
						long cpu_reg_count = (1 << (global_region - cpu_region)) + 1; 		//Número de regiões 			      -	 +1 para a condição de parada incluir todos
						long cpu_pos_count = 1 << (cpu_region - 1); 						//Número de posições na região 	-	 -1 porque são duas posições por iteração      

				
						long cpu_ext_proj_id = 0;
						long inc_ext_proj_id = ~(cpu_reg_mask ^ global_reg_mask) & ((1 << qubits) - 1);
			
						long proj_id;		//indentificador local da região
						proj_id = cpu_ext_proj_id | cpu_proj_id;
						cpu_ext_proj_id = (cpu_ext_proj_id + inc_ext_proj_id + 1) & ~inc_ext_proj_id;
						cpu_reg_count--;
						
						while (proj_id != -1){
							//Computa os operadores
							PCpuExecution1_0(state, pts, qubits, cpu_start, cpu_end, cpu_pos_count, proj_id, cpu_reg_mask);
				
							proj_id = cpu_ext_proj_id | cpu_proj_id;
							cpu_ext_proj_id = (cpu_ext_proj_id + inc_ext_proj_id + 1) & ~inc_ext_proj_id;
							cpu_reg_count--;
							if (cpu_reg_count <= 0)
								proj_id = -1;
						}
			
						cpu_start = cpu_end;
					}
		
					#pragma omp critical (global_teste)
					{
						cpu_proj_id = ext_proj_id;
						ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
						global_reg_count--;
							if (global_reg_count <= 0)
						cpu_proj_id = -1;
					}
				}
				
			}
			//#pragma omp section          //GPU EXECUTION
			else{
				long gpu_proj_id;
				
				#pragma omp critical (global_teste)
				{
					gpu_proj_id = ext_proj_id;
					ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
					global_reg_count--;
					if (global_reg_count <= 0)
						gpu_proj_id = -1;
				}

				while (gpu_proj_id != -1){
					//Project Gates
					vector <PT*> gpu_pts;
					
					int gpu_i;

					int map_qb[qubits];
					memset(map_qb, -1, qubits * sizeof(int));
		
					int m = 0;
					for (gpu_i = 0; gpu_i < qubits; gpu_i++){
						if ((1 << gpu_i) & global_reg_mask){
							map_qb[gpu_i] = m++;
						}
					}
					
					PT *aux;
					gpu_pts.clear();
					for (int gpu_i = global_start; gpu_i < global_end; gpu_i++){
						
						//verifica se o controle do operador satisfaz a parte global da região
						if ((pts[gpu_i]->ctrl_mask & gpu_proj_id & ~global_reg_mask) == (pts[gpu_i]->ctrl_value & ~global_reg_mask)){
							aux = new PT();

							aux->qubits = pts[gpu_i]->qubits;

							aux->matrix = pts[gpu_i]->matrix;
							aux->mat_size = pts[gpu_i]->mat_size;
							aux->ctrl_mask = pts[gpu_i]->ctrl_mask & global_reg_mask;
							aux->ctrl_value = pts[gpu_i]->ctrl_value & global_reg_mask;

							aux->end = map_qb[pts[gpu_i]->end];
							aux->start = aux->end - log2((float)aux->mat_size);

							aux->ctrl_count = 0;
							for (int c = global_coales; c < qubits; c++){
								if (aux->ctrl_mask & (1<<c)){
									aux->ctrl_count++;

									aux->ctrl_mask &= ~(1<<c);			//retira da mascara o controle do qubit atual (c)
									aux->ctrl_mask |= (1 << map_qb[c]);	//e coloca o qubit que ele mapeia (map_qb[c])

									if (aux->ctrl_value & (1<<c)){ 		//se o valor do controle for zero faz a mesma coisa para ctrl_value;
										aux->ctrl_mask &= ~(1<<c);
										aux->ctrl_mask |= (1 << map_qb[c]);
									}
								}
							}	

							gpu_pts.push_back(aux);
						}
					}
					gpu_pts.push_back(NULL);
					////////////////

					ProjectState(state, qubits, global_region, gpu_proj_id, global_reg_mask, multi_gpu);

					GpuExecutionWrapper(NULL, &gpu_pts[0], global_region, gpu_coales, gpu_region, multi_gpu, tam_block, rept, 1);
	
					GetState(state, qubits, global_region, gpu_proj_id, global_reg_mask, multi_gpu);

					for (int c = 0; c < gpu_pts.size() - 1; c++){
						delete gpu_pts[c];
					}
		
					#pragma omp critical (global_teste)
					{
						gpu_proj_id = ext_proj_id;
						ext_proj_id = (ext_proj_id + global_reg_mask + 1) & ~global_reg_mask;
						global_reg_count--;
						if (global_reg_count <= 0)
							gpu_proj_id = -1;
					}
				}
			}
			
		//}
		}
	}
}

void DGM::setCpuStructure(long cpu_region, long cpu_coales){
	this->cpu_region = cpu_region;
	this->cpu_coales = cpu_coales;
}

void DGM::setGpuStructure(long gpu_region, long gpu_coales, int rept){
	this->gpu_region = gpu_region;
	this->gpu_coales = gpu_coales;
	this->rept = rept;
	this->tam_block = 1 << gpu_region / 2 / rept;
}