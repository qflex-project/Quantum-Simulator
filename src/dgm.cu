#include "hip/hip_runtime.h"
#include <mpi.h>
#include <omp.h>
#include <math.h>
#include <unistd.h>

#include <cstdio>
#include <iostream>
#include <iterator>

#include "cpu.h"
#include "dcpu.h"
#include "dgm.h"
#include "gpu.h"
#include "hybrid.h"
#include "pcpu.h"

void Tokenize(const std::string &str, std::vector<std::string> &tokens,
              const std::string &delimiters = ",") {
  // Skip delimiters at beginning.
  std::string::size_type lastPos = str.find_first_not_of(delimiters, 0);
  // Find first "non-delimiter".
  std::string::size_type pos = str.find_first_of(delimiters, lastPos);

  while (std::string::npos != pos || std::string::npos != lastPos) {
    // Found a token, add it to the std::vector.
    tokens.push_back(str.substr(lastPos, pos - lastPos));
    // Skip delimiters.  Note the "not_of"
    lastPos = str.find_first_not_of(delimiters, pos);
    // Find next "non-delimiter"
    pos = str.find_first_of(delimiters, lastPos);
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////

float complex *GenericExecute(float complex *state, std::string function,
                              int qubits, int type, int threads,
                              int factor = 0) {
  DGM dgm;
  dgm.exec_type = type;
  dgm.cpu_params.n_threads = threads;
  dgm.qubits = qubits;
  dgm.factor = factor;

  dgm.setMemory(state);

  dgm.executeFunction(function);

  state = dgm.state;

  dgm.state = NULL;

  return state;
}

float complex *GenericExecute(float complex *state,
                              std::vector<std::string> function, int qubits,
                              int type, int threads, int factor = 0) {
  DGM dgm;
  dgm.exec_type = type;
  dgm.cpu_params.n_threads = threads;
  dgm.qubits = qubits;
  dgm.factor = factor;
  dgm.setMemory(state);

  dgm.executeFunction(function);

  dgm.state = NULL;

  return state;
}

///////////////////////////////////////////////////////////////////////////////////////////////

DGM::DGM() {
  MAX_QB = QB_LIMIT;
  MAX_PT = PT_TAM;

  pts = NULL;
  state = NULL;
  en_print = false;
  exec_type = t_CPU;
  factor = 1;
  gpu_params.multi_gpu = 1;
}

DGM::~DGM() { erase(); }

void DGM::setExecType(int type) { exec_type = type; }

void DGM::printPTs() {
  for (int i = 0; i < vec_pts.size() - 1; i++) {
    vec_pts[i]->print();
  }
}

void DGM::erase() {
  if (!pts) return;

  long i = 0;
  while (pts[i] != NULL) {
    pts[i]->destructor();
    free(pts[i]);
    i++;
  }

  vec_pts.clear();
  pts = NULL;
}

void DGM::allocateMemory() {
  state = (float complex *)calloc(pow(2, qubits), sizeof(float complex));
}

void DGM::setMemory(float complex *mem) {
  freeMemory();
  state = mem;
}

void DGM::freeMemory() {
  if (state) free(state);
  state = NULL;
}

void DGM::setMemoryValue(int pos) { state[pos] = 1; }

int DGM::measure(int q_pos) {
  long size = pow(2.0, qubits);

  long shift = (qubits - 1 - q_pos);

  int count_one, count_zero, num_pb;
  float zero, one, norm, r;
  one = zero = 0;

  // #pragma omp for;
  for (long i = 0; i < size; i++) {
    if ((i >> shift) & 1)
      one += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);
    else
      zero += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);
  }

  long m;
  srand(time(NULL));
  count_one = 0;
  count_zero = 0;
  num_pb = 1;

  for (int i = 0; i < num_pb; i++) {
    r = (double)rand() / RAND_MAX;
    if (zero > r)
      count_zero++;
    else
      count_one++;
  }

  if (count_one > count_zero) {
    measure_value = one;
    norm = sqrt(one);
    m = 1;
  } else {
    measure_value = zero;
    norm = sqrt(zero);
    m = 0;
  }

  long mask;
  mask = pow(2, shift) - 1;
#pragma omp for
  for (long i = 0; i < size / 2; i++) {
    long pos = (i << 1) - (i & mask);
    state[pos] = state[pos | (m << shift)] / norm;
    state[pos | (1 << shift)] = 0.0;
  }

  return m;
}

void DGM::colapse(int q_pos, int value) {
  long size = pow(2.0, qubits);
  long mask = (qubits - 1 - q_pos);

  float m;
  m = 0;

  for (long i = 0; i < size; i++)
    if (((i >> mask) & 1) == value)
      m += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);

  std::cout << m << std::endl;

  m = sqrt(m);
  for (long i = 0; i < size; i++) {
    if (((i >> mask) & 1) == value)
      state[i] = state[i] / m;
    else
      state[i] = 0.0;
  }
}

std::map<long, float> DGM::measure(std::vector<int> q_pos) {
  long mask = 0;

  for (int i = 0; i < q_pos.size(); i++)
    mask = mask | (1 << (qubits - 1 - q_pos[i]));

  std::map<long, float> m;

  long size = pow(2.0, qubits);

  for (long i = 0; i < size; i++)
    m[i & mask] += pow(crealf(state[i]), 2.0) + pow(cimagf(state[i]), 2.0);

  return m;
}

void DGM::setFunction(std::string function, int it, bool er) {
  std::vector<std::string> steps;

  Tokenize(function, steps, ";");

  setFunction(steps, it, er);
}

void DGM::setFunction(std::vector<std::string> steps, int it, bool er) {
  if (er)
    erase();
  else
    vec_pts.pop_back();

  std::vector<PT *> step_pts, vec_tmp;
  std::map<long, Group> gps;

  for (long j = 0; j < it; j++)
    for (long i = 0; i < steps.size(); i++) {
      gps = genGroups(steps[i]);
      genPTs(gps, step_pts);

      if (i % 2)
        sort(step_pts.begin(), step_pts.end(), increasing);
      else
        sort(step_pts.begin(), step_pts.end(), decreasing);

      vec_pts.insert(vec_pts.end(), step_pts.begin(), step_pts.end());
    }

  vec_pts.push_back(NULL);

  pts = &vec_pts[0];
}

std::map<long, Group> DGM::genGroups(std::string step) {
  std::vector<std::string> ops;
  Tokenize(step, ops);  // separa os operadores usando "," como delimitador
  qubits = ops.size();

  size_t found_c, found_t, p;
  std::string str;
  long pos, ctrl_value, ctrl_num;

  std::map<long, Group> gps;

  char *pEnd;
  pos = 0;
  std::vector<std::string>::iterator it;
  for (it = ops.begin(); it != ops.end(); ++it) {  // percorre os operadores
    str = *it;
    // std::cout << str << std::endl;
    found_c = str.find("Control");  // tamanho 7
    found_t = str.find("Target");   // tamanho 6
    p = str.find("(") + 1;

    if (found_c != std::string::npos) {  // Controle
      ctrl_num = strtol(str.c_str() + 7, &pEnd, 10);
      ctrl_value = strtol(str.c_str() + p, &pEnd, 10);

      gps[ctrl_num].ctrl.push_back(ctrl_value);  // adicona o valor do controle
      gps[ctrl_num].pos_ctrl.push_back(
          pos);  // e a sua posição ao map relacionado ao controle
    } else if (found_t != std::string::npos) {  // Target
      ctrl_num = strtol(str.c_str() + 6, &pEnd, 10);
      str = str.substr(p, str.size() - p - 1);

      gps[ctrl_num].ops.push_back(str);  // adicona o operador
      gps[ctrl_num].pos_ops.push_back(
          pos);           // e a sua posição ao map relacionado ao target
    } else {              // operador normal
      if (str != "ID") {  // se for ID ignora
        gps[0].ops.push_back(str);      // adiciona o operador
        gps[0].pos_ops.push_back(pos);  // e a sua posição ao map '0'
      }
    }
    pos++;
  }

  return gps;
}

void DGM::genPTs(std::map<long, Group> &gps, std::vector<PT *> &step_pts) {
  step_pts.clear();
  Gates gates;

  std::map<long, Group>::iterator it;
  Group gp;
  PT *pt;
  long ctrl_mask, ctrl_value, ctrl_count;
  long size;

  for (it = gps.begin(); it != gps.end(); ++it) {  // percorre os grupos
    gp = it->second;
    size = gp.ops.size();

    ctrl_count = gp.ctrl.size();
    ctrl_value = ctrl_mask = 0;

    for (long i = 0; i < ctrl_count;
         i++) {  // gera a mascara e o valor do controle (em binario)
      gp.pos_ctrl[i] = qubits - gp.pos_ctrl[i] - 1;
      ctrl_mask += (1 << gp.pos_ctrl[i]);
      if (gp.ctrl[i]) ctrl_value += (1 << gp.pos_ctrl[i]);
    }

    for (int p = 0; p < size; p++) {
      pt = (PT *)malloc(sizeof(PT));
      pt->affected = false;

      pt->qubits = 1;
      pt->start = qubits - gp.pos_ops[p];
      pt->end = pt->start - 1;
      pt->mat_size = 2;

      pt->matrix = gates.getMatrix(gp.ops[p]);

      pt->ctrl_value = ctrl_value;
      pt->ctrl_mask = ctrl_mask;
      pt->ctrl_count = ctrl_count;

      if (ctrl_count) {
        pt->ctrl_pos = (long *)malloc(sizeof(long) * ctrl_count);
        copy(gp.pos_ctrl.begin(), gp.pos_ctrl.end(), pt->ctrl_pos);
      }

      step_pts.push_back(pt);
    }
  }
}

void DGM::genMatrix(float complex *matrix,
                    std::vector<float complex *> &matrices, long tam,
                    long current, long line, long column, float complex cmplx) {
  if (cmplx == 0.0) return;

  if (current == tam) {  // percorreu até a ultima matriz
    matrix[line * (1 << tam) + column] = cmplx;
    return;
  }

  for (long l = 0; l < 2; l++)
    for (long c = 0; c < 2; c++)
      genMatrix(matrix, matrices, tam, current + 1, (line << 1) | l,
                (column << 1) | c, cmplx * matrices[current][l * 2 + c]);
}

void DGM::executeFunction(std::vector<std::string> function, int it) {
  setFunction(function);
  execute(it);
}

void DGM::executeFunction(std::string function, int it) {
  if (function == "") return;

  setFunction(function);
  execute(it);
}

float complex *DGM::execute(int it) {
  int initialized, finalized;

  MPI_Initialized(&initialized);
  if (!initialized) MPI_Init(NULL, NULL);
  float complex *result = state;

  int world_rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
  int world_size;
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);

  if (world_size > 1) {
    e_size region = qubits - log2((float)world_size);
    e_size coales = region - 2;
    e_size i = 0LL;
    e_size start = 0LL;
    e_size end = 0LL;
    while (pts[i] != NULL) {
      start = i;
      MaskNewRegion maskNewRegion = getMaskAndRegion(pts, coales, region, i);
      end = i;
    }
  } else {
    switch (exec_type) {
      case t_CPU:
        CpuExecution1(state, pts, qubits, it);
        break;
      case t_PAR_CPU:
        PCpuExecution1(state, pts, qubits, cpu_params.n_threads,
                       cpu_params.cpu_coales, cpu_params.cpu_region);
        break;
      case t_GPU:
        result = GpuExecutionWrapper(
            state, pts, qubits, gpu_params.gpu_coales, gpu_params.gpu_region,
            gpu_params.multi_gpu, gpu_params.tam_block, gpu_params.rept, it);
        break;
      case t_HYBRID:
        HybridExecution(state, pts, qubits, cpu_params, gpu_params);
        break;
      default:
        std::cout << "Erro exec type" << std::endl;
        exit(1);
    }
  }

  // not responsibility of this function
  // MPI_Finalized(&finalized);
  // if (!finalized) MPI_Finalize();

  return result;
}

OPSCounter DGM::CountOps(int it) {
  OPSCounter counter;

  for (int i = 0; pts[i] != NULL; i++) {
    long mt = pts[i]->matrixType();
    switch (mt) {
      case DENSE:
        (pts[i]->ctrl_mask) ? counter.c_dense++ : counter.dense++;
        break;
      case DIAG_PRI:
        (pts[i]->ctrl_mask) ? counter.c_main_diag++ : counter.main_diag++;
        break;
      case DIAG_SEC:
        (pts[i]->ctrl_mask) ? counter.c_sec_diag++ : counter.sec_diag++;
        break;
      default:
        std::cout << "Error on operator type" << std::endl;
        exit(1);
    }
  }

  counter.dense *= it;
  counter.c_dense *= it;
  counter.main_diag *= it;
  counter.c_main_diag *= it;
  counter.sec_diag *= it;
  counter.c_sec_diag *= it;

  counter.total_op = counter.dense + counter.c_dense + counter.main_diag +
                     counter.c_main_diag + counter.sec_diag +
                     counter.c_sec_diag;
  return counter;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void report_num_threads(int level) {
#pragma omp single
  {
    printf("Level %d: number of threads in the team - %d\n", level,
           omp_get_num_threads());
  }
}

void DGM::setCpuStructure(long cpu_region, long cpu_coales) {
  this->cpu_params.cpu_region = cpu_region;
  this->cpu_params.cpu_coales = cpu_coales;
}

void DGM::setGpuStructure(long gpu_region, long gpu_coales, int rept) {
  this->gpu_params.gpu_region = gpu_region;
  this->gpu_params.gpu_coales = gpu_coales;
  this->gpu_params.rept = rept;
  this->gpu_params.tam_block = 1 << gpu_region / 2 / rept;
}

// Coalescimento
void MPI_coalesc(float complex *state, int qubits, int proj_qubits, long reg_id,
                 long reg_mask, int world_size) {
  int qbs_coales = 0;
  for (int i = 0; i < qubits; i++) {
    if ((reg_mask >> i) & 1) {
      qbs_coales++;
    } else {
      break;
    }
  }

  int mem_portions = pow(2.0, proj_qubits - qbs_coales);
  int portion_size = 1 << qbs_coales;

  // float malloc_size = (1 << proj_qubits) * sizeof(float complex);

  float complex *new_state =
      (float complex *)(malloc(sizeof(float complex) * pow(2, qubits)));
  int *chunk_sizes = (int *)(malloc(sizeof(int) * world_size));
  int *displ = (int *)(malloc(sizeof(int) * world_size));

  long inc = ~(reg_mask >> qbs_coales);

  long dest_pos, src_pos, base = 0;
  for (int d = 0; d < world_size; d++) {
    displ[d] = dest_pos;
    for (int b = mem_portions / world_size * d;
         b < mem_portions / world_size * (d + 1); b++) {
      src_pos = (base << qbs_coales) | reg_id;

      memcpy(new_state + dest_pos, state + src_pos,
             portion_size * sizeof(float complex));

      base = (base + inc + 1) & ~inc;
      dest_pos += portion_size;
    }
    chunk_sizes[d] = dest_pos - displ[d];
  }
}
