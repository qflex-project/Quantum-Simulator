#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"


bool error();
static int inst = 0;
static int call_count = 0;
static int call_peer_count = 0;


struct host_partial_transform{
	PT *pt;
	PT *pts;
	int num_pts;
	hipFloatComplex *matrix, *read_memory, *write_memory;
	long read_size, write_size;

	void malloc(long size, long pt_size){
		hipMalloc((void**)&write_memory, size);
		hipMalloc((void**)&read_memory, size);
		hipMalloc((void**)&matrix, pow(4, pt_size)*sizeof(float complex));
	};
	void malloc(long r_size, long w_size, long pt_size){
		read_size = r_size;
		write_size = w_size;
		
		hipMalloc((void**)&read_memory, read_size*sizeof(float complex));
		hipMalloc((void**)&write_memory, write_size*sizeof(float complex));
		hipMalloc((void**)&matrix, pow(4, pt_size)*sizeof(float complex));
	};

	void malloc_read(long size, long pt_size){
		write_memory = NULL;
		//hipMalloc((void**)&write_memory, size);
		hipMalloc((void**)&read_memory, size);
		hipMalloc((void**)&matrix, pow(4, pt_size)*sizeof(float complex));	
	}

	void free(){
		if (write_memory) hipFree(write_memory);
		hipFree(read_memory);
		hipFree(matrix); 
	};
	
	void swap(){
		hipFloatComplex **ptr1, **ptr2;
		ptr1 = &read_memory;
		ptr2 = &write_memory;
		
		
		hipFloatComplex *aux = *ptr1;
		*ptr1 = *ptr2;
		*ptr2 = aux;
	}
};
typedef host_partial_transform HPT;

struct DEV_OP{
	long arg[TAM_ARG];
	hipFloatComplex matrix[4];
};


extern "C" bool setDevice(int num = 0){
	return hipFree(0);
}

extern "C" bool enablePeerAccess(){
	hipSetDevice(0);
	hipDeviceEnablePeerAccess(1, 0);
    
    hipSetDevice(1);
    hipDeviceEnablePeerAccess(0, 0);

    hipGetLastError();

    return true;
}

__constant__ long c_arg[1][1];
__constant__ hipFloatComplex cmatrix[1][1];

__constant__ DEV_OP op[OPS_BLOCK];

__constant__ hipFloatComplex *gpu_pointer[4];


inline int GET_BLOCK_ID(PT *pt, int coalesc, int qbs_region){
	return (pt->end - coalesc)/(qbs_region-coalesc);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//extern "C"
template <int t_TAM_BLOCK, int t_REPT, int t_COALESC>
__global__ void ApplyValuesC01(int const b_pos, int const n_bits, int const count, int const shift, int const block_shift){
	long pos, block = (blockIdx.x + block_shift) * t_REPT;
	
	int i, c, thId = threadIdx.x;

	__shared__ hipFloatComplex s[t_REPT][t_TAM_BLOCK*2];

	long block_base[t_REPT];

	for (i = 0; i < t_REPT; i++){
		block_base[i] = (block + i) << t_COALESC;
		block_base[i] = (block_base[i] >> b_pos) << (b_pos + n_bits) | (block_base[i] & ((1 << b_pos) - 1)); //OPEN SPACE
	}
	
	
	for (i = 0; i < t_REPT; i++){
		pos = block_base[i] | ((thId >> t_COALESC) << b_pos) | (thId & ((1 << t_COALESC)-1));
		s[i][thId] = gpu_pointer[pos/shift][pos%shift];

		pos = pos | (1 << (b_pos+n_bits-1));
		s[i][thId+t_TAM_BLOCK] = gpu_pointer[pos/shift][pos%shift];
	}



	int pos0, pos1, op_bit;	
	hipFloatComplex tmp;
	
	for (c = 0; c < count; c++){
		__syncthreads();

		//if (print) printf("Ctrl G: M -> %ld    V -> %ld\n", op[c].arg[CTRL_MASK], op[c].arg[CTRL_VALUE]);
		//if (print) printf("Ctrl R: M -> %ld    V -> %ld\n", op[c].arg[CTRL_REG_MASK], op[c].arg[CTRL_REG_VALUE]);
		op_bit = 1 << op[c].arg[SHIFT];

		pos0 = (thId * 2) - (thId & (op_bit - 1));
		pos1 = pos0 | op_bit;
		
		for (i = 0; i < t_REPT; i++){
			if (((block_base[i] & op[c].arg[CTRL_MASK]) == op[c].arg[CTRL_VALUE]) && ((pos0 & op[c].arg[CTRL_REG_MASK]) == op[c].arg[CTRL_REG_VALUE])){

				//long ctrl = block_base | ((pos0 >> COALESC) << b_pos) | (pos0 & ((1 << COALESC)-1));

					//if (print) printf("threadIdx: %d  - opbit: %d  ---    pos: %d    e   %d\n", threadIdx.x, op_bit, pos0, pos1);

					tmp = hipCaddf(hipCmulf(s[i][pos0], op[c].matrix[0]), hipCmulf(s[i][pos1], op[c].matrix[1]));
					s[i][pos1] = hipCaddf(hipCmulf(s[i][pos0], op[c].matrix[2]), hipCmulf(s[i][pos1], op[c].matrix[3]));			
					s[i][pos0] = tmp;
			}
		}
	}
	__syncthreads();


	for (i = 0; i < t_REPT; i++){
		pos = block_base[i] | ((thId >> t_COALESC) << b_pos) | (thId & ((1 << t_COALESC)-1));
		gpu_pointer[pos/shift][pos%shift] = s[i][thId];

		pos = pos | (1 << (b_pos+n_bits-1));
		gpu_pointer[pos/shift][pos%shift] = s[i][thId+t_TAM_BLOCK];
	}
	
}



template <int t_TAM_BLOCK, int t_REPT, int t_COALESC>	
void GpuExecution01(float* r_memory, PT **pts, int qubits, int qbs_region, int multi_gpu, int num_it){
	//printf("%d  --  %d  --  %d  --  %d\n", t_TAM_BLOCK, t_REPT, t_COALESC, qbs_region);
	DEV_OP operators[OPS_BLOCK];

	inst = 0;

	dim3 block, dim;

	long mem_size = pow(2.0, qubits);
	long mem_desloc = mem_size/multi_gpu;
	
	//long rept = REPT;		//número de substate cada bloco fica responsável
	long nth = mem_size/multi_gpu/t_REPT/2;//2;	// /2 porque cada thread fica responsável por duas posições & /2 pelas 2 GPUS

	long malloc_size = (mem_size * (sizeof(float complex)))/multi_gpu;

	
	block.x = t_TAM_BLOCK;
	(nth > block.x)? dim.x = nth/block.x : block.x = nth;


	int block_region_size = log(block.x)/log(2) + 1;

	if (block_region_size < qbs_region){
		printf("ERRO: Região do bloco menor que a região de qubits\n");
		exit(1);
	}

	hipFloatComplex *gpu_mem[4];


	if (multi_gpu > 1){
		for (int d = 0; d < multi_gpu; d++){
			hipSetDevice(d);
			for (int j = 0; j < multi_gpu; j++)
				if (d!=j) hipDeviceEnablePeerAccess(j, 0);
		}
		hipGetLastError();
	}
	

	for (int d = 0; d < multi_gpu; d++){
		hipSetDevice(d);
		hipMalloc(&gpu_mem[d], malloc_size); error();
		hipMemcpy(gpu_mem[d], r_memory + (mem_desloc*2)*d, malloc_size, hipMemcpyHostToDevice); error();
	}

	for (int d = 0; d < multi_gpu; d++){
		hipSetDevice(d);
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_pointer), gpu_mem, multi_gpu*sizeof(hipFloatComplex*)); error();
	}

	int i;
	for (int it = 0; it < num_it; it++){
		i = 0;

		while (pts[i]!= NULL){
			int qbs_block_id, region_start, is_peer, max_end, c = 0; //CONTADOR
			is_peer = 0;

			while (pts[i+c] != NULL &&
				   pts[i+c]->end < t_COALESC &&
				   c < OPS_BLOCK)
			{
				c++;
			}

			max_end = t_COALESC;

			if (pts[i+c] != NULL &&
				c < OPS_BLOCK)
			{
				qbs_block_id = GET_BLOCK_ID(pts[i+c], t_COALESC, qbs_region);

				do
				{
					max_end = max(max_end, pts[i+c]->end);
					c++;
				}
				while (pts[i+c] != NULL &&
					   qbs_block_id == GET_BLOCK_ID(pts[i+c], t_COALESC, qbs_region) &&
					   c < OPS_BLOCK);
			}

			region_start = max(t_COALESC, (max_end - (block_region_size - t_COALESC) + 1));

			is_peer = ((region_start + (block_region_size - t_COALESC)) > (qubits-multi_gpu+1));

			//printf("COUNT %d\nREGION_START %d\nREGION_BLOCK %d\nSHIFT %ld\nDIMX %d\n", c, region_start, (block_region_size - COALESC), mem_desloc, dim.x);
					

			for (int j = 0; j < c; j++){
				memcpy(operators[j].matrix, pts[i+j]->matrix, 4*sizeof(float complex));
				pts[i+j]->setArgsGPU(operators[j].arg, region_start, block_region_size, t_COALESC);
			}

			if (is_peer){
				for (int d = 0; d < multi_gpu; d++){
					hipSetDevice(d);
					hipDeviceSynchronize();
				}
			}

			for (int d = 0; d < multi_gpu; d++){
				hipSetDevice(d);
				hipMemcpyToSymbol(HIP_SYMBOL(op), operators, c*sizeof(DEV_OP));
			}
			
			for (int d = 0; d < multi_gpu; d++){
				hipSetDevice(d);
				ApplyValuesC01<t_TAM_BLOCK, t_REPT, t_COALESC><<<dim,block>>>(region_start, (block_region_size - t_COALESC), c, mem_desloc, dim.x * d);
			}

			
			if (is_peer){
				for (int d = 0; d < multi_gpu; d++){
					hipSetDevice(d);
					hipDeviceSynchronize();
				}
			}

			call_count++;
			if (is_peer) call_peer_count++;

			i += c;
		}
	}

	for (int d = 0; d < multi_gpu; d++){
		hipMemcpy(r_memory + (mem_desloc*2)*d, gpu_mem[d], malloc_size, hipMemcpyDeviceToHost); error();
		hipFree(gpu_mem[d]); error();
	}

	//printf("Kernel Calls %d\nPeer Calls %d\n", call_count, call_peer_count);
}



template <int t_COALESC>
void GEWrapper2(float* r_memory, PT **pts, int qubits, int qbs_region, int multi_gpu, int tam_block, int rept, int num_it){
	switch(tam_block){
		case 32:
			switch(rept){
				case 1:
					GpuExecution01<32, 1, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 2:
					GpuExecution01<32, 2, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 4:
					GpuExecution01<32, 4, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 8:
					GpuExecution01<32, 8, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 16:
					GpuExecution01<32, 16, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 32:
					GpuExecution01<32, 32, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				default:
					printf("Invalid REPT");
			}
			break;
		case 64:
			switch(rept){
				case 1:
					GpuExecution01<64, 1, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 2:
					GpuExecution01<64, 2, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 4:
					GpuExecution01<64, 4, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 8:
					GpuExecution01<64, 8, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 16:
					GpuExecution01<64, 16, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 32:
					GpuExecution01<64, 32, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				default:
					printf("Invalid REPT");
			}
			break;
		case 128:
			switch(rept){
				case 1:
					GpuExecution01<128, 1, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 2:
					GpuExecution01<128, 2, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 4:
					GpuExecution01<128, 4, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 8:
					GpuExecution01<128, 8, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 16:
					GpuExecution01<128, 16, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				default:
					printf("Invalid REPT");
			}
			break;
		case 256:
			switch(rept){
				case 1:
					GpuExecution01<256, 1, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 2:
					GpuExecution01<256, 2, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 4:
					GpuExecution01<256, 4, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 8:
					GpuExecution01<256, 8, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				default:
					printf("Invalid REPT");
			}
			break;
		case 512:
			switch(rept){
				case 1:
					GpuExecution01<512, 1, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 2:
					GpuExecution01<512, 2, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 4:
					GpuExecution01<512, 4, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				default:
					printf("Invalid REPT");
			}
			break;
		case 1024:
			switch(rept){
				case 1:
					GpuExecution01<1024, 1, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				case 2:
					GpuExecution01<1024, 2, t_COALESC>(r_memory, pts, qubits, qbs_region, multi_gpu, num_it);
					break;
				default:
					printf("Invalid REPT");
			}
			break;
		default:
			printf("Invalid TAM_BLOCK");
	}
}


extern "C" float* GpuExecutionWrapper(float* r_memory, PT **pts, int qubits, int qbs_region, int multi_gpu, int tam_block, int rept, int coalesc, int num_it){
	switch(coalesc){
		case 0:
			GEWrapper2<0>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 1:
			GEWrapper2<1>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 2:
			GEWrapper2<2>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 3:
			GEWrapper2<3>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 4:
			GEWrapper2<4>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 5:
			GEWrapper2<5>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 6:
			GEWrapper2<6>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 7:
			GEWrapper2<7>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		case 8:
			GEWrapper2<8>(r_memory, pts, qubits, qbs_region, multi_gpu, tam_block, rept, num_it);
			break;
		default:
			printf("Invalid COALESC");
	}

	return r_memory;
}

bool error(){
	inst++;
	hipError_t e;
	e = hipGetLastError();
	if (e == hipSuccess) return false;
	printf("inst: %d\nerror: %d - %s\n", inst, e, hipGetErrorString (e));
	exit(1);
	return true;
}
